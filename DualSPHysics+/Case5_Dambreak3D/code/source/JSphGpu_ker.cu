#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2020 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License 
 as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.
 
 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details. 

 You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpu_ker.cu \brief Implements functions and CUDA kernels for the Particle Interaction and System Update.

#include "JSphGpu_ker.h"
#include "Functions.h"
#include "FunctionsCuda.h"
#include "JLog2.h"
#include <cfloat>
#include <hip/hip_math_constants.h>
//:#include "JDgKerPrint.h"
//:#include "JDgKerPrint_ker.h"

#pragma warning(disable : 4267) //Cancels "warning C4267: conversion from 'size_t' to 'int', possible loss of data"
#pragma warning(disable : 4244) //Cancels "warning C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data"
#pragma warning(disable : 4503) //Cancels "warning C4503: decorated name length exceeded, name was truncated"
#include <thrust/device_vector.h>
#include <thrust/sort.h>

__constant__ StCteInteraction CTE;
#define CTE_AVAILABLE

namespace cusph{
#include "FunctionsBasic_iker.h"
#include "FunctionsMath_iker.h"
#include "FunctionsGeo3d_iker.h"
#include "FunSphKernel_iker.h"
#include "FunSphEos_iker.h"
#include "JCellSearch_iker.h"


//==============================================================================
/// Reduction using maximum of float values in shared memory for a warp.
/// Reduccion mediante maximo de valores float en memoria shared para un warp.
//==============================================================================
template <unsigned blockSize> __device__ void KerReduMaxFloatWarp(volatile float* sdat,unsigned tid){
  if(blockSize>=64)sdat[tid]=max(sdat[tid],sdat[tid+32]);
  if(blockSize>=32)sdat[tid]=max(sdat[tid],sdat[tid+16]);
  if(blockSize>=16)sdat[tid]=max(sdat[tid],sdat[tid+8]);
  if(blockSize>=8)sdat[tid]=max(sdat[tid],sdat[tid+4]);
  if(blockSize>=4)sdat[tid]=max(sdat[tid],sdat[tid+2]);
  if(blockSize>=2)sdat[tid]=max(sdat[tid],sdat[tid+1]);
}

//==============================================================================
/// Accumulates the maximum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula el maximo de n valores del vector dat[], guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat(unsigned n,unsigned ini,const float *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini]: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
///
/// Devuelve el maximo de un vector, usando resu[] como vector auxiliar. El tamanho
/// de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
//==============================================================================
float ReduMaxFloat(unsigned ndata,unsigned inidata,float* data,float* resu){
  float resf=0;
  if(ndata>=1){
    unsigned n=ndata,ini=inidata;
    unsigned smemSize=SPHBSIZE*sizeof(float);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    unsigned n_blocks=sgrid.x*sgrid.y;
    float *dat=data;
    float *resu1=resu,*resu2=resu+n_blocks;
    float *res=resu1;
    while(n>1){
      KerReduMaxFloat<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,dat,res);
      n=n_blocks; ini=0;
      sgrid=GetSimpleGridSize(n,SPHBSIZE);  
      n_blocks=sgrid.x*sgrid.y;
      if(n>1){
        dat=res; res=(dat==resu1? resu2: resu1); 
      }
    }
    if(ndata>1)hipMemcpy(&resf,res,sizeof(float),hipMemcpyDeviceToHost);
    else hipMemcpy(&resf,data,sizeof(float),hipMemcpyDeviceToHost);
  }
  //else{//-Using Thrust library is slower than ReduMasFloat() with ndata < 5M.
  //  thrust::device_ptr<float> dev_ptr(data);
  //  resf=thrust::reduce(dev_ptr,dev_ptr+ndata,-FLT_MAX,thrust::maximum<float>());
  //}
  return(resf);
}

//==============================================================================
/// Accumulates the sum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula la suma de n valores del vector dat[].w, guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat_w(unsigned n,unsigned ini,const float4 *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini].w: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
///
/// Devuelve el maximo de la componente w de un vector float4, usando resu[] como 
/// vector auxiliar. El tamanho de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
//==============================================================================
float ReduMaxFloat_w(unsigned ndata,unsigned inidata,float4* data,float* resu){
  unsigned n=ndata,ini=inidata;
  unsigned smemSize=SPHBSIZE*sizeof(float);
  dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
  unsigned n_blocks=sgrid.x*sgrid.y;
  float *dat=NULL;
  float *resu1=resu,*resu2=resu+n_blocks;
  float *res=resu1;
  while(n>1){
    if(!dat)KerReduMaxFloat_w<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,data,res);
    else KerReduMaxFloat<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,dat,res);
    n=n_blocks; ini=0;
    sgrid=GetSimpleGridSize(n,SPHBSIZE);  
    n_blocks=sgrid.x*sgrid.y;
    if(n>1){
      dat=res; res=(dat==resu1? resu2: resu1); 
    }
  }
  float resf;
  if(ndata>1)hipMemcpy(&resf,res,sizeof(float),hipMemcpyDeviceToHost);
  else{
    float4 resf4;
    hipMemcpy(&resf4,data,sizeof(float4),hipMemcpyDeviceToHost);
    resf=resf4.w;
  }
  return(resf);
}

//==============================================================================
/// Stores constants for the GPU interaction.
/// Graba constantes para la interaccion a la GPU.
//==============================================================================
void CteInteractionUp(const StCteInteraction *cte){
  hipMemcpyToSymbol(HIP_SYMBOL(CTE),cte,sizeof(StCteInteraction));
}

//------------------------------------------------------------------------------
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//------------------------------------------------------------------------------
__global__ void KerInitArray(unsigned n,float3 *v,float3 value)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p]=value;
}

//==============================================================================
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//==============================================================================
void InitArray(unsigned n,float3 *v,tfloat3 value){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInitArray <<<sgrid,SPHBSIZE>>> (n,v,Float3(value));
  }
}

//------------------------------------------------------------------------------
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//------------------------------------------------------------------------------
__global__ void KerResety(unsigned n,unsigned ini,float3 *v)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p+ini].y=0;
}

//==============================================================================
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//==============================================================================
void Resety(unsigned n,unsigned ini,float3 *v){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerResety <<<sgrid,SPHBSIZE>>> (n,ini,v);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float3 r=ace[p];
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace.
//==============================================================================
void ComputeAceMod(unsigned n,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,ace,acemod);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace, comprobando que la particula sea normal.
/// Uses zero for periodic particles.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const typecode rcod=code[p];
    const float3 r=(CODE_IsNormal(rcod) && !CODE_IsFluidInout(rcod)? ace[p]: make_float3(0,0,0));
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace, comprobando que la particula sea normal.
/// Uses zero for periodic particles.
//==============================================================================
void ComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,code,ace,acemod);
  }
}


//##############################################################################
//# Other kernels...
//# Otros kernels...
//##############################################################################
//------------------------------------------------------------------------------
/// Calculates module^2 of vel.
//------------------------------------------------------------------------------
__global__ void KerComputeVelMod(unsigned n,const float4 *vel,float *velmod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float4 r=vel[p];
    velmod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of vel.
//==============================================================================
void ComputeVelMod(unsigned n,const float4 *vel,float *velmod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeVelMod <<<sgrid,SPHBSIZE>>> (n,vel,velmod);
  }
}


//##############################################################################
//# Kernels para cambiar la posicion.
//# Kernels for changing the position.
//##############################################################################
//------------------------------------------------------------------------------
/// Updates pos, dcell and code from the indicated displacement.
/// The code may be CODE_OUTRHOP because in ComputeStepVerlet / Symplectic this is evaluated
/// and is executed before ComputeStepPos.
/// Checks limits depending on maprealposmin and maprealsize, this is valid 
/// for single-GPU because maprealpos and domrealpos are equal. For multi-gpu it is
/// important to mark particles that leave the domain without leaving the map.
///
/// Actualiza pos, dcell y code a partir del desplazamiento indicado.
/// Code puede ser CODE_OUTRHOP pq en ComputeStepVerlet/Symplectic se evalua esto 
/// y se ejecuta antes que ComputeStepPos.
/// Comprueba los limites en funcion de maprealposmin y maprealsize esto es valido
/// para single-gpu pq domrealpos y maprealpos son iguales. Para multi-gpu seria 
/// necesario marcar las particulas q salgan del dominio sin salir del mapa.
//------------------------------------------------------------------------------
template<bool periactive> __device__ void KerUpdatePos
  (double2 rxy,double rz,double movx,double movy,double movz
  ,bool outrhop,unsigned p,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  //-Checks validity of displacement. | Comprueba validez del desplazamiento.
  const bool outmove=(fmaxf(fabsf(float(movx)),fmaxf(fabsf(float(movy)),fabsf(float(movz))))>CTE.movlimit);
  //-Applies diplacement.
  double3 rpos=make_double3(rxy.x,rxy.y,rz);
  rpos.x+=movx; rpos.y+=movy; rpos.z+=movz;
  if(rpos.y<0 && CTE.symmetry)rpos.y=-rpos.y; //<vs_syymmetry>
  //-Checks limits of real domain. | Comprueba limites del dominio reales.
  double dx=rpos.x-CTE.maprealposminx;
  double dy=rpos.y-CTE.maprealposminy;
  double dz=rpos.z-CTE.maprealposminz;
  bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  if(periactive && out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    bool outx=!xperi && (dx<0 || dx>=CTE.maprealsizex);
    bool outy=!yperi && (dy<0 || dy>=CTE.maprealsizey);
    bool outz=!zperi && (dz<0 || dz>=CTE.maprealsizez);
    out=(outx||outy||outz);
    rpos=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  //-Stores updated position.
  posxy[p]=make_double2(rpos.x,rpos.y);
  posz[p]=rpos.z;
  //-Stores cell and check. | Guarda celda y check.
  if(outrhop || outmove || out){//-Particle out. Only brands as excluded normal particles (not periodic). | Particle out. Solo las particulas normales (no periodicas) se pueden marcar como excluidas.
    typecode rcode=code[p];
    if(out)rcode=CODE_SetOutPos(rcode);
    else if(outrhop)rcode=CODE_SetOutRhop(rcode);
    else rcode=CODE_SetOutMove(rcode);
    code[p]=rcode;
    dcell[p]=DCEL_CodeMapOut;
  }
  else{//-Particle in.
    if(periactive){
      dx=rpos.x-CTE.domposminx;
      dy=rpos.y-CTE.domposminy;
      dz=rpos.z-CTE.domposminz;
    }
    const unsigned cx=unsigned(dx/CTE.scell);
    const unsigned cy=unsigned(dy/CTE.scell);
    const unsigned cz=unsigned(dz/CTE.scell);
    dcell[p]=DCEL_Cell(CTE.cellcode,cx,cy,cz);
  }
}

//------------------------------------------------------------------------------
/// Returns the corrected position after applying periodic conditions.
/// Devuelve la posicion corregida tras aplicar condiciones periodicas.
//------------------------------------------------------------------------------
__device__ double3 KerUpdatePeriodicPos(double3 ps)
{
  double dx=ps.x-CTE.maprealposminx;
  double dy=ps.y-CTE.maprealposminy;
  double dz=ps.z-CTE.maprealposminz;
  const bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  //-Adjusts position according to periodic conditions and rechecks domain limits.
  //-Ajusta posicion segun condiciones periodicas y vuelve a comprobar los limites del dominio.
  if(out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    ps=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  return(ps);
}


//##############################################################################
//# Kernels for calculating forces (Pos-Double).
//# Kernels para calculo de fuerzas (Pos-Double).
//##############################################################################
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. Bound-Fluid/Float
/// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool symm>
  __device__ void KerInteractionForcesBoundBox
  (unsigned p1,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned* idp
  ,float massf,const float4 &pscellp1,const float4 &velrhop1,float &arp1,float &visc)
{
  for(int p2=pini;p2<pfin;p2++){
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(PSCEL_GetfX(pscellp1.w)-PSCEL_GetfX(pscellp2.w));
    float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(PSCEL_GetfY(pscellp1.w)-PSCEL_GetfY(pscellp2.w));
    float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(PSCEL_GetfZ(pscellp1.w)-PSCEL_GetfZ(pscellp2.w));
    if(symm)dry=pscellp1.y+pscellp2.y + CTE.poscellsize*PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO){
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>
      //-Obtains particle mass p2 if there are floating bodies.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massf si es fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        bool ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massf);
        compute=!(USE_FTEXTERNAL && ftp2); //-Deactivated when DEM or Chrono is used and is bound-float. | Se desactiva cuando se usa DEM o Chrono y es bound-float.
      }

      if(compute){
        //-Density derivative (Continuity equation).
        const float dvx=velrhop1.x-velrhop2.x, dvy=velrhop1.y-velrhop2.y, dvz=velrhop1.z-velrhop2.z;
        arp1+=(USE_FLOATING? ftmassp2: massf)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

        {//===== Viscosity ===== 
          const float dot=drx*dvx + dry*dvy + drz*dvz;
          const float dot_rr2=dot/(rr2+CTE.eta2);
          visc=max(dot_rr2,visc); 
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Particle interaction. Bound-Fluid/Float
/// Realiza interaccion entre particulas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool symm> 
  __global__ void KerInteractionForcesBound(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid,const unsigned *dcell
  ,const float *ftomassp
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of thread.
  if(p<n){
    const unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0;

    //-Loads particle p1 data.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const bool rsymp1=(symm && PSCEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>
    
    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Boundary-Fluid interaction.
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin){
                          KerInteractionForcesBoundBox<tker,ftmode,false> (p1,pini,pfin,ftomassp,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc);
        if(symm && rsymp1)KerInteractionForcesBoundBox<tker,ftmode,true > (p1,pini,pfin,ftomassp,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc);
      }
    }
    //-Stores results.
    if(arp1 || visc){
      ar[p1]+=arp1;
      if(visc>viscdt[p1])viscdt[p1]=visc;
    }
  }
}


  //##############################################################################
  //------------------------------------------------------------------------------
  /// Interaction of a particle with a set of particles. Bound-Fluid/Float
  /// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
  //------------------------------------------------------------------------------
  template<TpKernel tker, TpFtMode ftmode, bool symm>
  __device__ void KerInteractionForcesBoundDummyBox
  (unsigned p1, const unsigned& pini, const unsigned& pfin, const float* ftomassp
      , const float4* poscell, const float4* velrhop, const typecode* code, const unsigned* idp
      , float massf, const float4& pscellp1, const float4& velrhop1, float& arp1, float& visc, float& pressp1
      , float& kernelsum)
  {
      for (int p2 = pini; p2 < pfin; p2++) {
          const float4 pscellp2 = poscell[p2];
          float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
          float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
          float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
          if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
          const float rr2 = drx * drx + dry * dry + drz * drz;
          if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
              //-Computes kernel.
              const float wab = cufsph::GetKernel_Wab<tker>(rr2);
              //pressp1 += (p);
              kernelsum += wab;
              const float fac = cufsph::GetKernel_Fac<tker>(rr2);
              const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
              float4 velrhop2 = velrhop[p2];
              if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
              //-Obtains particle mass p2 if there are floating bodies.
              //-Obtiene masa de particula p2 en caso de existir floatings.
              float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massf si es fluid.
              bool compute = true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
              if (USE_FLOATING) {
                  const typecode cod = code[p2];
                  bool ftp2 = CODE_IsFloating(cod);
                  ftmassp2 = (ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massf);
                  compute = !(USE_FTEXTERNAL && ftp2); //-Deactivated when DEM or Chrono is used and is bound-float. | Se desactiva cuando se usa DEM o Chrono y es bound-float.
              }

              if (compute) {
                  float pressp2 = cufsph::ComputePressCte(velrhop2.w);
                  pressp2 *= (pressp2 >= 0.0f);
                  //pressp1 += (pressp2 + (CTE.gravity.x * drx + CTE.gravity.y * dry + CTE.gravity.z * drz) * velrhop2.w) * wab;
                  float pressp2top1 = (pressp2 + (CTE.gravity.x * drx + CTE.gravity.y * dry + CTE.gravity.z * drz) * velrhop2.w) * wab;
                  pressp1 += fmaxf(pressp2top1, 0.0f);
                  //-Density derivative (Continuity equation).
                  const float dvx = velrhop1.x - velrhop2.x, dvy = velrhop1.y - velrhop2.y, dvz = velrhop1.z - velrhop2.z;
                  //arp1 += (USE_FLOATING ? ftmassp2 : massf) * (dvx * frx + dvy * fry + dvz * frz) * (velrhop1.w / velrhop2.w);
                  arp1 = 0;

                  {//===== Viscosity ===== 
                      const float dot = drx * dvx + dry * dvy + drz * dvz;
                      const float dot_rr2 = dot / (rr2 + CTE.eta2);
                      visc = max(dot_rr2, visc);
                  }
              }
          }
      }
  }

//------------------------------------------------------------------------------
/// Particle interaction. Bound-Fluid/Float
/// Realiza interaccion entre particulas. Bound-Fluid/Float
//------------------------------------------------------------------------------
  template<TpKernel tker, TpFtMode ftmode, bool symm>
  __global__ void KerInteractionForcesBoundDummy(unsigned n, unsigned pinit
      , int scelldiv, int4 nc, int3 cellzero, const int2* beginendcellfluid, const unsigned* dcell
      , const float* ftomassp, const float4* poscell, float4* velrhop, const typecode* code, const unsigned* idp
      , float* viscdt, float* ar, float* psi)
  {
      const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of thread.
      if (p < n) {
          const unsigned p1 = p + pinit;      //-Number of particle.
          float visc = 0, arp1 = 0;
          float pressP1 = 0, kernelSum = 0;

          //-Loads particle p1 data.
          const float4 pscellp1 = poscell[p1];
          const float4 velrhop1 = velrhop[p1];
          const bool rsymp1 = (symm && PSCEL_GetPartY(__float_as_uint(pscellp1.w)) == 0); //<vs_syymmetry>

          //-Obtains neighborhood search limits.
          int ini1, fin1, ini2, fin2, ini3, fin3;
          cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

          //-Boundary-Fluid interaction.
          for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
              unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, beginendcellfluid, pini, pfin);
              if (pfin) {
                  KerInteractionForcesBoundDummyBox<tker, ftmode, false>(p1, pini, pfin, ftomassp, poscell, velrhop, code, idp, CTE.massf, pscellp1, velrhop1, arp1, visc, pressP1, kernelSum);
                  if (symm && rsymp1)KerInteractionForcesBoundDummyBox<tker, ftmode, true >(p1, pini, pfin, ftomassp, poscell, velrhop, code, idp, CTE.massf, pscellp1, velrhop1, arp1, visc, pressP1, kernelSum);
              }
          }
          if (kernelSum) {
              pressP1 = pressP1 / kernelSum;
              velrhop[p1].w = cufsph::ComputeRhoCte(pressP1);
          }
          else velrhop[p1].w = CTE.rhopzero;

          //if(psi)psi[p1] = 0;//-Dirichlet Boundary conKerInteractionForcesFluidBoxditions for PSI is recommended
          //-Stores results.
          if (arp1 || visc) {
              ar[p1] += arp1;
              if (visc > viscdt[p1])viscdt[p1] = visc;
          }
      }
  }

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool lamsps,TpDensity tdensity,bool shift,bool symm>
  __device__ void KerInteractionForcesFluidBox(bool boundp2,unsigned p1
  ,const unsigned &pini,const unsigned &pfin,float visco
  ,const float *ftomassp,const float2 *tauff,const float3 *dengradcorr
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float3 *normalvec
  ,float massp2,bool ftp1
  ,const float4 &pscellp1,const float4 &velrhop1,float pressp1
  ,const float2 &taup1_xx_xy,const float2 &taup1_xz_yy,const float2 &taup1_yz_zz
  ,float2 &grap1_xx_xy,float2 &grap1_xz_yy,float2 &grap1_yz_zz
  ,float3 &acep1,float &arp1,float &visc,float &deltap1
  ,TpShifting shiftmode,float4 &shiftposfsp1,const unsigned &pardetp1
  ,bool hpdc, const float& psip1, const float* psi, float& apsip1)
{
  for(int p2=pini;p2<pfin;p2++){
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(PSCEL_GetfX(pscellp1.w)-PSCEL_GetfX(pscellp2.w));
    float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(PSCEL_GetfY(pscellp1.w)-PSCEL_GetfY(pscellp2.w));
    float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(PSCEL_GetfZ(pscellp1.w)-PSCEL_GetfZ(pscellp2.w));
    if(symm)dry=pscellp1.y+pscellp2.y + CTE.poscellsize*PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO){
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 if any floating bodies exist.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massp2);
        #ifdef DELTA_HEAVYFLOATING
          if(ftp2 && tdensity==DDT_DDT && ftmassp2<=(massp2*1.2f))deltap1=FLT_MAX;
        #else
          if(ftp2 && tdensity==DDT_DDT)deltap1=FLT_MAX;
        #endif
        if(ftp2 && shift && shiftmode==SHIFT_NoBound)shiftposfsp1.x=FLT_MAX; //-Cancels shifting with floating bodies. | Con floatings anula shifting.
        compute=!(USE_FTEXTERNAL && ftp1 && (boundp2 || ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>
            
      //-Velocity derivative (Momentum equation).
      //if(compute){
      //  const float pressp2=cufsph::ComputePressCte(velrhop2.w);
      //  const float prs=(pressp1+pressp2)/(velrhop1.w*velrhop2.w)
      //    +(tker==KERNEL_Cubic? cufsph::GetKernelCubic_Tensil(rr2,velrhop1.w,pressp1,velrhop2.w,pressp2): 0);
      //  const float p_vpm=-prs*(USE_FLOATING? ftmassp2: massp2);
      //  acep1.x+=p_vpm*frx; acep1.y+=p_vpm*fry; acep1.z+=p_vpm*frz;
      //}
      
      ////-One-side riemann sph problem
      float pressp2 = cufsph::ComputePressCte(velrhop2.w);
      float rr = sqrt(rr2);
      float3 eij = make_float3(-drx/rr, -dry/rr, -drz/rr);
      float3 normalvecp2 = normalvec[p2];
      float ul = (velrhop1.x * eij.x + velrhop1.y * eij.y + velrhop1.z * eij.z);
      float ur = (velrhop2.x * eij.x + velrhop2.y * eij.y + velrhop2.z * eij.z);//Zhang JCP2017
      float pl = pressp1;
      float pr = pressp2;
      if(boundp2){
          ul = -(normalvecp2.x * velrhop1.x + normalvecp2.y * velrhop1.y + normalvecp2.z * velrhop1.z);
          ur = -ul - 2.0f * (normalvecp2.x * velrhop2.x + normalvecp2.y * velrhop2.y + normalvecp2.z * velrhop2.z);
          pl = pressp1;
          pr = (pressp1 + velrhop1.w * (CTE.gravity.x * -drx + CTE.gravity.y * -dry + CTE.gravity.z * -drz));
          pr = pr *(pr>0.0f);
          velrhop2.w = cufsph::ComputeRhoCte(pr);
      }

      float beta0 = CTE.beta;
      if (compute) {
          //  //-Velocity derivative (Momentum equation).
          const float phi0 = beta0 * CTE.kernelh / rr * fminf(fmaxf(ul - ur, 0.0f), CTE.cs0);
          const float limiter = 0.5f * phi0 * (velrhop1.w + velrhop2.w) / 2.0f * (ul - ur);
          bool tic = (pl <= 0.0f && pardetp1 == 4);
          const float Pstar = tic ? 0.5f * (pr - pl) + limiter : 0.5f * (pr + pl) + limiter;
          //const float Pstar = 0.5f * (pr + pl) + limiter;
          const float p_vpm = -2.0f * massp2 * Pstar / (velrhop1.w * velrhop2.w);
          acep1.x += p_vpm * frx; acep1.y += p_vpm * fry; acep1.z += p_vpm * frz;
      }

      //-HPDC
      if (hpdc) {
          const float psi_vpm = -(psi[p2] - psip1) * massp2 / velrhop2.w;
          acep1.x += psi_vpm * frx; acep1.y += psi_vpm * fry; acep1.z += psi_vpm * frz;
          const float dvx = velrhop1.x - velrhop2.x, dvy = velrhop1.y - velrhop2.y, dvz = velrhop1.z - velrhop2.z;
          //apsip1 += (velrhop1.x * velrhop1.x + velrhop1.y * velrhop1.y + velrhop1.z * velrhop1.z)* massp2 * (dvx * frx + dvy * fry + dvz * frz) / velrhop2.w;
          apsip1 += CTE.cs0 * CTE.cs0 * massp2 * (dvx * frx + dvy * fry + dvz * frz) / velrhop2.w;
          //printf("apsip1 = %.3f, p1 = %d\n", apsip1,p1);
      }

      //-Density derivative (Continuity equation).
      const float dvx=velrhop1.x-velrhop2.x, dvy=velrhop1.y-velrhop2.y, dvz=velrhop1.z-velrhop2.z;
      if(compute)arp1+=(USE_FLOATING? ftmassp2: massp2)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

      const float cbar=CTE.cs0;
      const float dot3=(tdensity!=DDT_None || shift? drx*frx+dry*fry+drz*frz: 0);
      //-Density Diffusion Term (Molteni and Colagrossi 2009).
      if(tdensity==DDT_DDT && deltap1!=FLT_MAX){
        const float rhop1over2=velrhop1.w/velrhop2.w;
        const float visc_densi=CTE.ddtkh*cbar*(rhop1over2-1.f)/(rr2+CTE.eta2);
        const float delta=visc_densi*dot3*(USE_FLOATING? ftmassp2: massp2);
        //deltap1=(boundp2? FLT_MAX: deltap1+delta);
        //deltap1=(boundp2 && CTE.tboundary==BC_DBC? FLT_MAX: deltap1+delta);
        deltap1 = deltap1 + delta;
      }
      //-Density Diffusion Term (Fourtakas et al 2019).
      if((tdensity==DDT_DDT2 || (tdensity==DDT_DDT2Full && !boundp2)) && deltap1!=FLT_MAX && !ftp2){
        const float rh=1.f+CTE.ddtgz*drz;
        const float drhop=CTE.rhopzero*pow(rh,1.f/CTE.gamma)-CTE.rhopzero;  
        const float visc_densi=CTE.ddtkh*cbar*((velrhop2.w-velrhop1.w)-drhop)/(rr2+CTE.eta2);
        const float delta=visc_densi*dot3*massp2/velrhop2.w;
        //deltap1=(boundp2? FLT_MAX: deltap1-delta); //-blocks it makes it boil - bloody DBC
        deltap1 = deltap1 - delta;
      }

      //-Shifting correction.
      //if(shift && shiftposfsp1.x!=FLT_MAX){
      //  const float massrhop=(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
      //  const bool noshift=(boundp2 && (shiftmode==SHIFT_NoBound || (shiftmode==SHIFT_NoFixed && CODE_IsFixed(code[p2]))));
      //  shiftposfsp1.x=(noshift? FLT_MAX: shiftposfsp1.x+massrhop*frx); //-Removes shifting for the boundaries. | Con boundary anula shifting.
      //  shiftposfsp1.y+=massrhop*fry;
      //  shiftposfsp1.z+=massrhop*frz;
      //  shiftposfsp1.w-=massrhop*dot3;
      //}

      //===== Viscosity ===== 
      if(compute){
        const float dot=drx*dvx + dry*dvy + drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        if(!lamsps){//-Artificial viscosity.
          if(dot<0){
            const float amubar=CTE.kernelh*dot_rr2;  //amubar=CTE.kernelh*dot/(rr2+CTE.eta2);
            const float robar=(velrhop1.w+velrhop2.w)*0.5f;
            const float pi_visc=(-visco*cbar*amubar/robar)*(USE_FLOATING? ftmassp2: massp2);
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
        else{//-Laminar+SPS viscosity.
          {//-Laminar contribution.
            const float robar2=(velrhop1.w+velrhop2.w);
            const float temp=4.f*visco/((rr2+CTE.eta2)*robar2);  //-Simplication of temp=2.0f*visco/((rr2+CTE.eta2)*robar); robar=(rhopp1+velrhop2.w)*0.5f;
            const float vtemp=(USE_FLOATING? ftmassp2: massp2)*temp*(drx*frx+dry*fry+drz*frz);  
            acep1.x+=vtemp*dvx; acep1.y+=vtemp*dvy; acep1.z+=vtemp*dvz;
          }
          //-SPS turbulence model.
          float2 taup2_xx_xy=taup1_xx_xy; //-taup1 is always zero when p1 is not fluid. | taup1 siempre es cero cuando p1 no es fluid.
          float2 taup2_xz_yy=taup1_xz_yy;
          float2 taup2_yz_zz=taup1_yz_zz;
          if(!boundp2 && (USE_NOFLOATING || !ftp2)){//-When p2 is fluid.
            float2 taup2=tauff[p2*3];     taup2_xx_xy.x+=taup2.x; taup2_xx_xy.y+=taup2.y;
                   taup2=tauff[p2*3+1];   taup2_xz_yy.x+=taup2.x; taup2_xz_yy.y+=taup2.y;
                   taup2=tauff[p2*3+2];   taup2_yz_zz.x+=taup2.x; taup2_yz_zz.y+=taup2.y;
          }
          acep1.x+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xx_xy.x*frx+taup2_xx_xy.y*fry+taup2_xz_yy.x*frz);
          acep1.y+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xx_xy.y*frx+taup2_xz_yy.y*fry+taup2_yz_zz.x*frz);
          acep1.z+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xz_yy.x*frx+taup2_yz_zz.x*fry+taup2_yz_zz.y*frz);
          //-Velocity gradients.
          if(USE_NOFLOATING || !ftp1){//-When p1 is fluid.
            const float volp2=-(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
            float dv=dvx*volp2; grap1_xx_xy.x+=dv*frx; grap1_xx_xy.y+=dv*fry; grap1_xz_yy.x+=dv*frz;
                  dv=dvy*volp2; grap1_xx_xy.y+=dv*frx; grap1_xz_yy.y+=dv*fry; grap1_yz_zz.x+=dv*frz;
                  dv=dvz*volp2; grap1_xz_yy.x+=dv*frx; grap1_yz_zz.x+=dv*fry; grap1_yz_zz.y+=dv*frz;
            // to compute tau terms we assume that gradvel.xy=gradvel.dudy+gradvel.dvdx, gradvel.xz=gradvel.dudz+gradvel.dwdx, gradvel.yz=gradvel.dvdz+gradvel.dwdy
            // so only 6 elements are needed instead of 3x3.
          }
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool lamsps,bool hpdc,TpDensity tdensity,bool shift,bool symm>
  __global__ void KerInteractionForcesFluid(unsigned n,unsigned pinit,float viscob,float viscof
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,const float2 *tauff,float2 *gradvelff,const float3 *dengradcorr
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar,float3 *ace,float *delta
  ,TpShifting shiftmode,float4 *shiftposfs
  ,const unsigned *pardet, const float3 *normalvec,const float *psi, float *apsi)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0,deltap1=0;
    float3 acep1=make_float3(0,0,0);

    //-Variables for HPDC
    float psip1 = 0, apsip1 = 0;
    if(hpdc)psip1 = psi[p1];

    //-Variables for Shifting.
    float4 shiftposfsp1;
    if(shift)shiftposfsp1=shiftposfs[p1];

    //-Particle type
    const unsigned pardetp1 = pardet[p1];

    //-Obtains data of particle p1 in case there are floating bodies.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    if(USE_FLOATING){
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
      if(ftp1 && tdensity!=DDT_None)deltap1=FLT_MAX; //-DDT is not applied to floating particles.
      if(ftp1 && shift)shiftposfsp1.x=FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const float pressp1=cufsph::ComputePressCte(velrhop1.w);
    const bool rsymp1=(symm && PSCEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Variables for Laminar+SPS.
    float2 taup1_xx_xy,taup1_xz_yy,taup1_yz_zz;
    if(lamsps){
      taup1_xx_xy=tauff[p1*3];
      taup1_xz_yy=tauff[p1*3+1];
      taup1_yz_zz=tauff[p1*3+2];
    }
    //-Variables for Laminar+SPS (computation).
    float2 grap1_xx_xy,grap1_xz_yy,grap1_yz_zz;
    if(lamsps){
      grap1_xx_xy=make_float2(0,0);
      grap1_xz_yy=make_float2(0,0);
      grap1_yz_zz=make_float2(0,0);
    }

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
                          KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,false> (false,p1,pini,pfin,viscof,ftomassp,tauff,dengradcorr,poscell,velrhop,code,idp,normalvec,CTE.massf,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,pardetp1,hpdc, psip1, psi, apsip1);
        if(symm && rsymp1)KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,true > (false,p1,pini,pfin,viscof,ftomassp,tauff,dengradcorr,poscell,velrhop,code,idp,normalvec,CTE.massf,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,pardetp1,hpdc, psip1, psi, apsip1); //<vs_syymmetry>
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
                        KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,false> (true ,p1,pini,pfin,viscob,ftomassp,tauff,NULL,poscell,velrhop,code,idp,normalvec,CTE.massb,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,pardetp1,hpdc, psip1, psi, apsip1);
      if(symm && rsymp1)KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,true > (true ,p1,pini,pfin,viscob,ftomassp,tauff,NULL,poscell,velrhop,code,idp,normalvec,CTE.massb,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,pardetp1,hpdc, psip1, psi, apsip1);
      }
    }

    //-Compute dpsi/dt =======================================
    if (hpdc) {
        //const float _tau = sqrtf(velrhop1.x*velrhop1.x+velrhop1.y*velrhop1.y+velrhop1.z*velrhop1.z) * PI / (2.0f * CTE.kernelh);
        const float _tau = CTE.cs0 * PI / (2.0f * CTE.kernelh);
        apsip1 += -_tau * psip1;
    }

    //-Stores results.
    if(shift||arp1||acep1.x||acep1.y||acep1.z||visc){
      if(tdensity!=DDT_None){
        if(delta){
          const float rdelta=delta[p1];
          delta[p1]=(rdelta==FLT_MAX || deltap1==FLT_MAX? FLT_MAX: rdelta+deltap1);
        }
        else if(deltap1!=FLT_MAX)arp1+=deltap1;
      }
      ar[p1]+=arp1;
      if (hpdc)apsi[p1] += apsip1;
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      if(visc>viscdt[p1])viscdt[p1]=visc;
      if(lamsps){
        float2 rg;
        rg=gradvelff[p1*3  ];  rg=make_float2(rg.x+grap1_xx_xy.x,rg.y+grap1_xx_xy.y);  gradvelff[p1*3  ]=rg;
        rg=gradvelff[p1*3+1];  rg=make_float2(rg.x+grap1_xz_yy.x,rg.y+grap1_xz_yy.y);  gradvelff[p1*3+1]=rg;
        rg=gradvelff[p1*3+2];  rg=make_float2(rg.x+grap1_yz_zz.x,rg.y+grap1_yz_zz.y);  gradvelff[p1*3+2]=rg;
      }
      //if(shift)shiftposfs[p1]=shiftposfsp1;
    }
  }
}

#ifndef DISABLE_BSMODES
//==============================================================================
/// Collects kernel information.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,bool lamsps,bool hpdc,TpDensity tdensity,bool shift,bool symm> 
  void Interaction_ForcesT_KerInfo(StKerInfo *kerinfo)
{
 #if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,unsigned,float,float,int,int4,int3,const int2*,unsigned,const unsigned*,const float*,const float2*,float2*,const float3*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*,float3*,float*,TpShifting,float4*,const unsigned *, const float3 *,const float *, float *);
    fun_ptr ptr=&KerInteractionForcesFluid<tker,ftmode,lamsps,hpdc,tdensity,shift,symm>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesfluid_bs=qblocksize;
    kerinfo->forcesfluid_rg=attr.numRegs;
    kerinfo->forcesfluid_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesFluid  blocksize:%u (%u)\n",qblocksize,0);
  }
  {
    typedef void (*fun_ptr)(unsigned,unsigned,int,int4,int3,const int2*,const unsigned*,const float*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*);
    fun_ptr ptr=&KerInteractionForcesBound<tker,ftmode,symm>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesbound_bs=qblocksize;
    kerinfo->forcesbound_rg=attr.numRegs;
    kerinfo->forcesbound_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesBound  blocksize:%u (%u)\n",qblocksize,0);
  }
  fcuda::Check_CudaErroorFun("Error collecting kernel information.");
 #endif
}
#endif


//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,bool lamsps,bool hpdc,TpDensity tdensity,bool shift> 
  void Interaction_ForcesGpuT(const StInterParmsg &t)
{
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(t.kerinfo){
    Interaction_ForcesT_KerInfo<tker,ftmode,lamsps,hpdc,tdensity,shift,false>(t.kerinfo);
    return;
  }
#endif
  const StDivDataGpu &dvd=t.divdatag;
  const int2* beginendcell=dvd.beginendcell;

  //-Interaction Boundary-Fluid.
  if (t.boundnum) {
      const int2* beginendcellfluid = dvd.beginendcell + dvd.cellfluid;
      dim3 sgridb = GetSimpleGridSize(t.boundnum, t.bsbound);
      if (t.symmetry) //<vs_syymmetry_ini>
          KerInteractionForcesBoundDummy<tker, ftmode, true > << <sgridb, t.bsbound, 0, t.stm >> >
          (t.boundnum, t.boundini, dvd.scelldiv, dvd.nc, dvd.cellzero, beginendcell + dvd.cellfluid, t.dcell
              , t.ftomassp, t.poscell, t.velrhop, t.code, t.idp, t.viscdt, t.ar, t.psi);
      else //<vs_syymmetry_end>
          KerInteractionForcesBoundDummy<tker, ftmode, false> << <sgridb, t.bsbound, 0, t.stm >> >
          (t.boundnum, t.boundini, dvd.scelldiv, dvd.nc, dvd.cellzero, beginendcellfluid, t.dcell
              , t.ftomassp, t.poscell, t.velrhop, t.code, t.idp, t.viscdt, t.ar, t.psi);
  }

  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(t.fluidnum){
    //printf("[ns:%u  id:%d] halo:%d fini:%d(%d) bini:%d(%d)\n",t.nstep,t.id,t.halo,t.fluidini,t.fluidnum,t.boundini,t.boundnum);
    dim3 sgridf=GetSimpleGridSize(t.fluidnum,t.bsfluid);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesFluid<tker,ftmode,lamsps,hpdc,tdensity,shift,true> <<<sgridf,t.bsfluid,0,t.stm>>> 
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.ftomassp,(const float2*)t.tau,(float2*)t.gradvel,t.dengradcorr,t.poscell,t.velrhop,t.code,t.idp
      ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs,t.pardet,t.normalvec, t.psi, t.apsi);
    else //<vs_syymmetry_end>
      KerInteractionForcesFluid<tker,ftmode,lamsps,hpdc,tdensity,shift,false> <<<sgridf,t.bsfluid,0,t.stm>>> 
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.ftomassp,(const float2*)t.tau,(float2*)t.gradvel,t.dengradcorr,t.poscell,t.velrhop,t.code,t.idp
      ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs,t.pardet,t.normalvec, t.psi, t.apsi);
  }
}

//==============================================================================
//#define FAST_COMPILATION
template<TpKernel tker,TpFtMode ftmode,bool lamsps, bool hpdc> void Interaction_Forces_gt3(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.shiftmode || t.tdensity!=DDT_DDT4)throw "Shifting and extra DDT are disabled for FastCompilation...";
  Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT4,false> (t);
#else
  if(t.shiftmode){               const bool shift=true;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_None    ,shift> (t);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT     ,shift> (t);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT2    ,shift> (t);
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT2Full,shift> (t);
  }
  else{                           const bool shift=false;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_None    ,shift> (t);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT     ,shift> (t);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT2    ,shift> (t);
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT<tker,ftmode,lamsps,hpdc,DDT_DDT2Full,shift> (t);
  }
#endif
}


//#define FAST_COMPILATION
template<TpKernel tker, TpFtMode ftmode, bool lamsps> void Interaction_Forces_gt2(const StInterParmsg& t) {
    if (t.tdiv == DIV_VEM_HPDC)    Interaction_Forces_gt3<tker, ftmode, lamsps, true>(t);
    else                           Interaction_Forces_gt3<tker, ftmode, lamsps, false>(t);
}

//==============================================================================
template<TpKernel tker,TpFtMode ftmode> void Interaction_Forces_gt1(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.lamsps)throw "Extra viscosity options are disabled for FastCompilation...";
  Interaction_Forces_gt2<tker,ftmode,false> (t);
#else
  if(t.lamsps)Interaction_Forces_gt2<tker,ftmode,true>  (t);
  else        Interaction_Forces_gt2<tker,ftmode,false> (t);
#endif
}
//==============================================================================
template<TpKernel tker> void Interaction_Forces_gt0(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.ftmode!=FTMODE_None)throw "Extra FtMode options are disabled for FastCompilation...";
  Interaction_Forces_gt1<tker,FTMODE_None> (t);
#else
  if(t.ftmode==FTMODE_None)    Interaction_Forces_gt1<tker,FTMODE_None> (t);
  else if(t.ftmode==FTMODE_Sph)Interaction_Forces_gt1<tker,FTMODE_Sph>  (t);
  else if(t.ftmode==FTMODE_Ext)Interaction_Forces_gt1<tker,FTMODE_Ext>  (t);
#endif
}
//==============================================================================
void Interaction_Forces(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.tkernel!=KERNEL_Wendland)throw "Extra kernels are disabled for FastCompilation...";
  Interaction_Forces_gt0<KERNEL_Wendland> (t);
#else
  if(t.tkernel==KERNEL_Wendland)     Interaction_Forces_gt0<KERNEL_Wendland> (t);
 #ifndef DISABLE_KERNELS_EXTRA
  else if(t.tkernel==KERNEL_Cubic)   Interaction_Forces_gt0<KERNEL_Cubic   > (t);
 #endif
#endif
}

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<TpKernel tker>
__device__ void KerGetAuxarrayBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop
    , float massp2, const float4& pscellp1, const float4& velrhop1, tmatrix3f& kera_corr, float &cp1, float& posdivp1, unsigned& np2ofp1, float & veldivp1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.

            float4 velrhop2 = velrhop[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = massp2 / velrhop2.w;
            //-corrective matrix
            kera_corr.a11 += -drx * frx * vb; kera_corr.a12 += -drx * fry * vb; kera_corr.a13 += -drx * frz * vb;
            kera_corr.a21 += -dry * frx * vb; kera_corr.a22 += -dry * fry * vb; kera_corr.a23 += -dry * frz * vb;
            kera_corr.a31 += -drz * frx * vb; kera_corr.a32 += -drz * fry * vb; kera_corr.a33 += -drz * frz * vb;
            //-ci
            cp1 += wab * vb;
            //-Divergence of position vec
            posdivp1 += -(drx * frx + dry * fry + drz * frz) * vb;
            np2ofp1 += 1;
            //-Divergence of velocity
            const float dvx = velrhop1.x - velrhop2.x, dvy = velrhop1.y - velrhop2.y, dvz = velrhop1.z - velrhop2.z;
            veldivp1 += -vb * (dvx * frx + dvy * fry + dvz * frz);
        }
    }
}

//------------------------------------------------------------------------------
template<TpKernel tker>
__global__ void KerGetAuxarray(bool simulate2d,unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, tmatrix3f* corrmat,unsigned *pardet, float* veldiv, float *ci) {

    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];

        //-Corrective matrix
        tmatrix3f a_corr;cumath::Tmatrix3fReset(a_corr);
        //-Ci
        float cp1 = cufsph::GetKernel_Wab<tker>(0.0f) * CTE.massf / velrhop1.w;//-add p1
        //-Div of position vector
        float posdivp1 = 0.0f;//-divergence of position vector
        unsigned np2ofp1 = 0.0f;//-number of neighbor particles of p1 
        float lowerlimit = simulate2d ? 0.4f : 0.6f, upperlimit = simulate2d ? 1.75f : 2.5f;//divergence of position vector lower and upper limits
        float nlimit = 0.4f;//ratio : (p2 of p1)/n0
        //-Div u
        float veldivp1 = 0.0f;
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);
        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetAuxarrayBox<tker>(p1, pini, pfin, poscell, velrhop, CTE.massf, pscellp1, velrhop1, a_corr, cp1, posdivp1, np2ofp1, veldivp1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetAuxarrayBox<tker>(p1, pini, pfin, poscell, velrhop, CTE.massb, pscellp1, velrhop1, a_corr, cp1, posdivp1, np2ofp1, veldivp1);
            }
        }

        //-Get Corrective Matrix
        const double determ = simulate2d ? cumath::Determinant2x2(a_corr) :cumath::Determinant3x3(a_corr);
        if (determ != 0) {
            tmatrix3f corrmatp1 = (simulate2d ? cumath::InverseMatrix2x2(a_corr, determ) : cumath::InverseMatrix3x3(a_corr, determ));
            corrmat[p1] = corrmatp1;
        }
        else {
            cumath::Tmatrix3fResetI(corrmat[p1]);
        }
        //-Get Ci
        ci[p1] = cp1;

        //-Get particle detection step I
        if (posdivp1 < lowerlimit) pardet[p1] = 1;//-splash particles
        else if (posdivp1 >= lowerlimit && posdivp1 < upperlimit)pardet[p1] = 2;//-free surface and vicinitity particles
        else pardet[p1] = 4;//inner paticles
        //pardet[p1] = 4;
        //if (posdivp1 < lowerlimit || np2ofp1 < nlimit* CTE.nzero) pardet[p1] = 1;//-splash particles
        //else if (posdivp1 >= lowerlimit && posdivp1 < upperlimit && np2ofp1 >= nlimit*CTE.nzero)pardet[p1] = 2;//-free surface and vicinitity particles
        
        //-Get Velocity Divergence
        veldiv[p1] = veldivp1;
    }
}


//------------------------------------------------------------------------------
template<TpKernel tker>
__global__ void KerGetAuxarrayBound(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, tmatrix3f* corrmat, unsigned* pardet, float* ci) {

    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p+pinit;
        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];

        //-corrective matrix
        tmatrix3f a_corr; cumath::Tmatrix3fReset(a_corr);
        //-Ci
        float cp1 = cufsph::GetKernel_Wab<tker>(0) * CTE.massb / velrhop1.w;//-add p1
        //-Div of position vector
        float posdivp1 = 0;//-divergence of position vector
        unsigned np2ofp1 = 0;//-number of neighbor particles of p1 
        float veldivp1 = 0;
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetAuxarrayBox<tker>(p1, pini, pfin, poscell, velrhop, CTE.massb, pscellp1, velrhop1, a_corr, cp1, posdivp1, np2ofp1,veldivp1);
            }
        }

        //-Get Corr Matrix
        const double determ = simulate2d ? cumath::Determinant2x2(a_corr) : cumath::Determinant3x3(a_corr);
        if (determ != 0) {
            tmatrix3f corrmatp1 = (simulate2d ? cumath::InverseMatrix2x2(a_corr, determ) : cumath::InverseMatrix3x3(a_corr, determ));
            corrmat[p1] = corrmatp1;
        }
        else {
            //printf("error in corrmat");
            cumath::Tmatrix3fResetI(corrmat[p1]);
        }
        //-Get Ci
        ci[p1] = cp1;
    }
}


//-----------------------------------------------------
//-----------------KerParachuteScan--------------------
//-----------------------------------------------------
template<TpKernel tker>
__device__ void KerParachuteScanBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, float massp2, const float4& pscellp1, const float4& velrhop1
    , const float3& normalvecp1, bool &isfsp1) {

    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            
            bool bool1 = false, bool2 = false;
            const float kernelh = CTE.kernelh;
            const float3 vec_rab = make_float3(-drx,-dry,-drz); 
            const float  cos_rab = (vec_rab.x * normalvecp1.x + vec_rab.y * normalvecp1.y + vec_rab.z * normalvecp1.z)/sqrt(rr2);
            const float3 vec_rtb = make_float3(-drx- kernelh* normalvecp1.x, -dry - kernelh * normalvecp1.y, -drz - kernelh * normalvecp1.z);
            
            const float  len_rtb2 = vec_rtb.x* vec_rtb.x + vec_rtb.y* vec_rtb.y + vec_rtb.z* vec_rtb.z;

            if(rr2 < 2.0f* kernelh * kernelh && cos_rab>0.75f) bool1 = true;
            if(rr2 >= 2.0f * kernelh * kernelh && len_rtb2< kernelh * kernelh) bool2=true;

            if(bool1 || bool2) {isfsp1=false;break;}

        }
    }

}

template<TpKernel tker>
__global__ void KerParachuteScan(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, float3* normalvec, unsigned* pardet) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle
        if(pardet[p1]==2){
            //-Obtains basic data of particle p1.
            const float4 pscellp1 = poscell[p1];
            const float4 velrhop1 = velrhop[p1];
            const float3 normalvecp1 = normalvec[p1];
            bool isfsp1 = true;
            //-Obtains neighborhood search limits.
            int ini1, fin1, ini2, fin2, ini3, fin3;
            cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

            //-Interaction with fluids.
            ini3 += cellfluid; fin3 += cellfluid;
            for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
                unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
                if (pfin) {
                    KerParachuteScanBox<tker>(p1, pini, pfin, poscell, velrhop, pardet, CTE.massf, pscellp1, velrhop1, normalvecp1,isfsp1);
                }
            }

            if(!isfsp1)pardet[p1]=4;

        }
    }
}

//-----------------------------------------------------
//--------------KerFreeSurfaceVicinity-----------------
//-----------------------------------------------------
template<TpKernel tker>
__device__ void KerFreeSurfaceVicinityBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, float massp2, const float4& pscellp1, const float4& velrhop1
    , bool& isfsvp1) {

    for (int p2 = pini; p2 < pfin; p2++) {
        if(pardet[p2]==2){
            const float4 pscellp2 = poscell[p2];
            float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
            float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
            float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
            //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
            const float rr2 = drx * drx + dry * dry + drz * drz;
            if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
                const float dismin = 2.0f*CTE.kernelh-0.5f*CTE.dp;
                if(rr2< dismin*dismin) {isfsvp1=true;break;}
            }
        }
    }
}

template<TpKernel tker>
__global__ void KerFreeSurfaceVicinity(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, unsigned* pardet) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle
        if (pardet[p1] == 4) {
            //-Obtains basic data of particle p1.
            const float4 pscellp1 = poscell[p1];
            const float4 velrhop1 = velrhop[p1];
            bool isfsvp1 = false;
            //-Obtains neighborhood search limits.
            int ini1, fin1, ini2, fin2, ini3, fin3;
            cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

            //-Interaction with fluids.
            ini3 += cellfluid; fin3 += cellfluid;
            for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
                unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
                if (pfin) {
                    KerFreeSurfaceVicinityBox<tker>(p1, pini, pfin, poscell, velrhop, pardet, CTE.massf, pscellp1, velrhop1, isfsvp1);
                }
            }

            if (!isfsvp1)pardet[p1] = 4;
            else pardet[p1]=3;
            //-bound vicinity?
        }
    }
}


//-----------------------------------------------------
//-------------------KerWallVicinity-------------------
//-----------------------------------------------------
template<TpKernel tker>
__device__ void KerWallVicinityBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, float massp2, const float4& pscellp1, const float4& velrhop1
    , bool& iswallvp1) {

    for (int p2 = pini; p2 < pfin; p2++) {
        if (pardet[p2] == 0) {
            const float4 pscellp2 = poscell[p2];
            float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
            float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
            float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
            //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
            const float rr2 = drx * drx + dry * dry + drz * drz;
            if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
                const float dismin = 1.8f * CTE.dp;
                if (rr2 < dismin * dismin) { iswallvp1 = true; break; }
            }
        }
    }
}

template<TpKernel tker>
__global__ void KerWallVicinity(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, unsigned* pardet) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle
        if (pardet[p1] == 2 || pardet[p1] == 3) {
            //-Obtains basic data of particle p1.
            const float4 pscellp1 = poscell[p1];
            const float4 velrhop1 = velrhop[p1];
            bool iswallvp1 = false;
            //-Obtains neighborhood search limits.
            int ini1, fin1, ini2, fin2, ini3, fin3;
            cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

            //-Interaction with fluids.
            //ini3 += cellfluid; fin3 += cellfluid;
            for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
                unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
                if (pfin) {
                    KerWallVicinityBox<tker>(p1, pini, pfin, poscell, velrhop, pardet, CTE.massf, pscellp1, velrhop1, iswallvp1);
                }
            }

            if (iswallvp1)pardet[p1] = 5;
        }
    }
}



//==============================================================================
///==================================VCS STEP ==================================
//==============================================================================

template<TpKernel tker>
__device__ void KerRunVcsPressStepIBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const float* pressvcs, const unsigned *pardet
    , float massp2, const float4& pscellp1, const float4& velrhop1, const unsigned& pardetp1,float& rhobarp1
    , float& aijp1, float& paijp1, float& dismininnerp1, unsigned& indexmininnerp1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            float4 velrhop2 = velrhop[p2];
            const float rhopp1 = velrhop1.w, rhopp2 = velrhop2.w;
            const float pvcsp2 = pressvcs[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = massp2 / rhopp2;
            // Rhobar sum
            rhobarp1 += wab * massp2;
            // Aij sum
            aijp1 += 8.0f * massp2 / (rhopp1 + rhopp2) / (rhopp1 + rhopp2)
                * (-1.0f) * (drx * frx + dry * fry + drz * frz) / rr2;
            // PAij sum
            paijp1 += 8.0f * massp2 / (rhopp1 + rhopp2) / (rhopp1 + rhopp2)
                * (-1.0f) * (drx * frx + dry * fry + drz * frz) / rr2 * pvcsp2;

            //-minimun distance to free surface (only for free-surface vicinity particles)
            if(pardetp1==3 && pardet[p2]==4){
                float dis = sqrt(rr2);
                dismininnerp1 = (dismininnerp1 > dis ? dis : dismininnerp1);
                indexmininnerp1 = (dismininnerp1 == dis? p2:indexmininnerp1);
            }
        }
    }
}

template<TpKernel tker>
__global__ void KerRunVcsPressStepI(unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const unsigned *pardet, const float* pressvcs
    , float* aij, float* paij, float* rhobari, float* pressvcsnewi,float* dismininner,unsigned *indexmininner,const double dt){
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        float rhobarp1 = cufsph::GetKernel_Wab<tker>(0) * CTE.massf;//-add p1
        float aijp1 = 0, paijp1 = 0, dismininnerp1 = FLT_MAX; unsigned indexmininnerp1 = 0;
        const unsigned pardetp1 = pardet[p1];
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVcsPressStepIBox<tker>(p1, pini, pfin, poscell, velrhop, pressvcs, pardet,CTE.massf, pscellp1, velrhop1,pardetp1, rhobarp1, aijp1, paijp1, dismininnerp1, indexmininnerp1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVcsPressStepIBox<tker>(p1, pini, pfin, poscell, velrhop, pressvcs, pardet, CTE.massf, pscellp1, velrhop1, pardetp1, rhobarp1, aijp1, paijp1, dismininnerp1, indexmininnerp1);
            }
        }
        aij[p1] = aijp1; paij[p1] = paijp1; rhobari[p1] = rhobarp1; dismininner[p1] = dismininnerp1; indexmininner[p1] = indexmininnerp1;
    }

    if(p<n){
        const unsigned p1 = p + pinit;      //-Number of particle.
        const float cs0 = CTE.cs0;
        const float rhobar0 = CTE.rhobarzero;
        const float rhopbysump1 = rhobari[p1];
        const float divcs = 1.0f / (rhobar0 * dt * dt * cs0 * cs0);
        const float eivcs = -(rhobar0 - rhopbysump1) / (rhobar0 * dt * dt);
        const float pressfin = (paij[p1] + eivcs) / (aij[p1] + divcs);
        pressvcsnewi[p1] = pressfin;
    }
}


template<TpKernel tker>
__global__ void KerRunVcsPress_I(unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, float* pressvcs
    , float* pressvcsnewi, float* dismininner, unsigned* indexmininner) {

    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;
        if (pardet[p1] == 1 || pardet[p1] == 2 || pardet[p1] == 5)pressvcsnewi[p1] = 0;
        //if (pardet[p1] == 3 && dismininner[p1] != FLT_MAX) {
        if (pardet[p1] == 3) {
            pressvcsnewi[p1] = pressvcsnewi[indexmininner[p1]]
                * (2.0f * CTE.kernelh - dismininner[p1]) / (2.0f * CTE.kernelh);
        }
        //if (pardet[p1] == 3 && dismininner[p1] == FLT_MAX) {
        //    pressvcsnewi[p1] = 0;
        //}
        pressvcs[p1] = pressvcsnewi[p1];
        //printf("cip1 = {%d,%d, %7.6f}\n",
        //    p1, pardet[p1],pressvcs[p1]);
    }
}

//##############################################################################
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. Bound-Fluid/Float
/// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker>
__device__ void KerRunVcsPress_IIBox
(unsigned p1, const unsigned& pini, const unsigned& pfin, const float4* poscell, const float4* velrhop, const float* pressvcs
    , float massf, const float4& pscellp1, const float4& velrhop1, float& pressvcsp1, float& kernelsum)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);

            kernelsum += wab;
            float4 velrhop2 = velrhop[p2];
            float pressvcsp2 = pressvcs[p2];
            //pressvcsp1 += pressvcsp2 * wab;
            //pressvcsp2 *= (pressvcsp2 >= 0.0f);
            pressvcsp1 += fmaxf(pressvcsp2*wab, 0.0f);
        }
    }
}


template<TpKernel tker>
__global__ void KerRunVcsPress_II(unsigned n, unsigned pinit, int scelldiv, int4 nc, int3 cellzero, const int2* begincell
    , unsigned cellfluid, const unsigned* dcell, const float4* poscell, const float4* velrhop, float* pressvcs) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        float pressvcsp1 = 0.0f, kernelsum = 0.0f;

        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);
        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVcsPress_IIBox<tker>(p1, pini, pfin, poscell, velrhop, pressvcs, CTE.massf, pscellp1, velrhop1, pressvcsp1, kernelsum);
            }
        }

        if(kernelsum)pressvcsp1 = pressvcsp1/kernelsum;
        pressvcs[p1] = pressvcsp1;
    }
}


//-------------VCS Shift Vector--------------

template<TpKernel tker>
__device__ void KerRunVcsShiftBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const float* pressvcs, float massp2
    , const float4& pscellp1, const float4& velrhop1,const tmatrix3f& corrmatp1
    , float3& shiftvcsp1, const float& pressvcsp1, const double dt)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            float4 velrhop2 = velrhop[p2];
            const float rhopp1 = velrhop1.w, rhopp2 = velrhop2.w;
            const float pvcsp2 = pressvcs[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = massp2 / rhopp2;
            float pconst = -dt * dt / rhopp1 * (pressvcs[p2] - pressvcsp1) * vb;//---------------------------------------------
            //double pconst = -dt / rhopp1 * (pressvcs[p2] - pressvcsp1) * vb;
            shiftvcsp1.x += (pconst * (corrmatp1.a11 * frx + corrmatp1.a12 * fry + corrmatp1.a13 * frz));
            shiftvcsp1.y += (pconst * (corrmatp1.a21 * frx + corrmatp1.a22 * fry + corrmatp1.a23 * frz));
            shiftvcsp1.z += (pconst * (corrmatp1.a31 * frx + corrmatp1.a32 * fry + corrmatp1.a33 * frz));
        }
    }
}

template<TpKernel tker>
__global__ void KerRunVcsShift(unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, const float* pressvcs
    , const tmatrix3f* corrmat, float3* shiftvcs,const double dt) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        const unsigned pardetp1 = pardet[p1];
        const tmatrix3f corrmatp1 = corrmat[p1];
        const float pressvcsp1 = pressvcs[p1];
        float3 shiftvcsp1 = make_float3(0, 0, 0);
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVcsShiftBox<tker>(p1, pini, pfin, poscell, velrhop, pressvcs, CTE.massf, pscellp1, velrhop1, corrmatp1, shiftvcsp1, pressvcsp1, dt);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVcsShiftBox<tker>(p1, pini, pfin, poscell, velrhop, pressvcs, CTE.massf, pscellp1, velrhop1, corrmatp1, shiftvcsp1, pressvcsp1, dt);
            }
        }

        //const float maxdist = CTE.kernelh/20.0f;
        ////const double shiftvcsnorm= sqrt(shiftvcsp1.x * shiftvcsp1.x + shiftvcsp1.y * shiftvcsp1.y + shiftvcsp1.z * shiftvcsp1.z);
        //shiftvcsp1.x = (fabs(shiftvcsp1.x) < maxdist ? shiftvcsp1.x : (shiftvcsp1.x >= 0 ? maxdist : -maxdist));
        //shiftvcsp1.y = (fabs(shiftvcsp1.y) < maxdist ? shiftvcsp1.y : (shiftvcsp1.y >= 0 ? maxdist : -maxdist));
        //shiftvcsp1.z = (fabs(shiftvcsp1.z) < maxdist ? shiftvcsp1.z : (shiftvcsp1.z >= 0 ? maxdist : -maxdist));
        shiftvcs[p1] = (pardetp1==1)? make_float3(0.0f,0.0f,0.0f) : shiftvcsp1;
    }
}



template<TpKernel tker>
__device__ void KerGetNormalsBox(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const float* ci
    , float massp2, const float4& pscellp1, const float4& velrhop1, const tmatrix3f& corrmatp1, float3& nablacp1, const float& cp1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
            float4 velrhop2 = velrhop[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = float(massp2) / float(velrhop2.w);
            nablacp1.x += (ci[p2] - cp1) * (corrmatp1.a11 * frx + corrmatp1.a12 * fry + corrmatp1.a13 * frz) * vb;
            nablacp1.y += (ci[p2] - cp1) * (corrmatp1.a21 * frx + corrmatp1.a22 * fry + corrmatp1.a23 * frz) * vb;
            nablacp1.z += (ci[p2] - cp1) * (corrmatp1.a31 * frx + corrmatp1.a32 * fry + corrmatp1.a33 * frz) * vb;
        }
    }
}

template<TpKernel tker>
__global__ void KerGetNormals(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const tmatrix3f* corrmat, float3* normalvec, float* ci) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.

    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const tmatrix3f  corrmatp1 = corrmat[p1];//-correction matrix p1
        //float3 normalvecp1 = normalvec[p1];//-normal vector p1
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        const float cp1 = ci[p1];
        float3 nablacp1 = make_float3(0, 0, 0);
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);
        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetNormalsBox<tker>(p1, pini, pfin, poscell, velrhop, ci, CTE.massf, pscellp1, velrhop1, corrmatp1, nablacp1, cp1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                //printf("gogogo");
                KerGetNormalsBox<tker>(p1, pini, pfin, poscell, velrhop, ci, CTE.massb, pscellp1, velrhop1, corrmatp1, nablacp1, cp1);
            }
        }

        const float vecnorm = sqrt(nablacp1.x * nablacp1.x + nablacp1.y * nablacp1.y + nablacp1.z * nablacp1.z);
        normalvec[p1] = make_float3(-nablacp1.x / vecnorm, -nablacp1.y / vecnorm, -nablacp1.z / vecnorm);
    }

}


template<TpKernel tker>
__global__ void KerGetNormalsBound(bool simulate2d, unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const tmatrix3f* corrmat, float3* normalvec, float* ci) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.

    if (p < n) {
        float3 nablacp1 = make_float3(0, 0, 0);
        const unsigned p1 = p+pinit;
        //const unsigned p1 = p ;      //-Number of particle.
        const tmatrix3f  corrmatp1 = corrmat[p1];//-correction matrix p1
        //float3 normalvecp1 = normalvec[p1];//-normal vector p1
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        const float cp1 = ci[p1];
            
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);
        //-Interaction with boundaries.
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                //printf("gogogo");
                KerGetNormalsBox<tker>(p1, pini, pfin, poscell, velrhop, ci, CTE.massb, pscellp1, velrhop1, corrmatp1, nablacp1, cp1);
            }
        }
        const float vecnorm = sqrt(nablacp1.x * nablacp1.x + nablacp1.y * nablacp1.y + nablacp1.z * nablacp1.z);
        normalvec[p1] = make_float3(-nablacp1.x / vecnorm, -nablacp1.y / vecnorm, -nablacp1.z / vecnorm);
    }

}


//==============================================================================
///==================================VCS STEP ==================================
//==============================================================================

template<TpKernel tker, bool vem, bool vcs, bool ops>
__device__ void KerRunVEMVCSOPS_StepIBox(bool boundp2, unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const float* pressvcs, const unsigned* pardet, const float3 *normalvec
    , float massp2, const float4& pscellp1, const float4& velrhop1, const unsigned& pardetp1, float &arp1, float& rhobarp1
    , float& aijp1, float& paijp1, float& dismininnerp1, unsigned& indexmininnerp1
    , const float3 normalvecp1, float& maxarccosp1, float& disminfsp1, unsigned& disminfsindexp1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            float4 velrhop2 = velrhop[p2];
            const float rhopp1 = velrhop1.w, rhopp2 = velrhop2.w;
            
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = massp2 / rhopp2;

            if(vem)
            {
                //------------------VEM------------------
                if(!boundp2){
                    const float dvx = velrhop1.x - velrhop2.x, dvy = velrhop1.y - velrhop2.y, dvz = velrhop1.z - velrhop2.z;
                    arp1 += massp2 * (dvx * frx + dvy * fry + dvz * frz) / velrhop2.w * rhopp1;
                }
            }

            if(vcs)
            {
                //------------------VCS------------------
                const float pvcsp2 = pressvcs[p2];
                // Rhobar sum
                rhobarp1 += wab * massp2;
                // Aij sum
                aijp1 += 8.0f * massp2 / (rhopp1 + rhopp2) / (rhopp1 + rhopp2)
                    * (-1.0f) * (drx * frx + dry * fry + drz * frz) / rr2;
                // PAij sum
                paijp1 += 8.0f * massp2 / (rhopp1 + rhopp2) / (rhopp1 + rhopp2)
                    * (-1.0f) * (drx * frx + dry * fry + drz * frz) / rr2 * pvcsp2;

                //-minimum distance to inner particles (only for free-surface vicinity particles)
                if (pardetp1 == 3 && pardet[p2] == 4) {
                    float dis = sqrt(rr2);
                    dismininnerp1 = (dismininnerp1 > dis ? dis : dismininnerp1);
                    indexmininnerp1 = (dismininnerp1 == dis ? p2 : indexmininnerp1);
                }
            }

            if(ops){
                //-------------------OPS------------------
                if (pardetp1 == 3 && pardet[p2] == 2) {
                    const float dis = sqrt(rr2);
                    disminfsp1 = (disminfsp1 > dis ? dis : disminfsp1);
                    disminfsindexp1 = (disminfsp1 == dis ? p2 : disminfsindexp1);
                }
                if (pardetp1 == 2 && pardet[p2] == 2) {
                    const float arccosp1p2 = acosf(normalvecp1.x * normalvec[p2].x + normalvecp1.y * normalvec[p2].y + normalvecp1.z * normalvec[p2].z);
                    maxarccosp1 = (arccosp1p2 > maxarccosp1 ? arccosp1p2 : maxarccosp1);
                }
            }
        }
    }
}

//-VEM VCS OPS StepI: Compute -VEM: pressure, -VCS: pressure -OPS: max arccos, min distance of Free surface vicinity particles to free surface
template<TpKernel tker, bool vem, bool vcs, bool ops>
__global__ void KerRunVEMVCSOPS_StepI(unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell, const float4* poscell
    , const float4* velrhop, const unsigned* pardet, const float* pressvcs, float* pressvem, float4* shiftposfs, const float3* normalvec
    , float* aij, float* paij, float* rhobari, float* pressvcsnewi, float* dismininner, unsigned* indexmininner, const double dt
    , float* maxarccos, float* disminfs, unsigned* disminindexfs, float* psig) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        //-For VEM
        float arp1 = 0;
        //-For VCS
        float rhobarp1 = cufsph::GetKernel_Wab<tker>(0) * CTE.massf;//-add p1
        float aijp1 = 0, paijp1 = 0, dismininnerp1 = FLT_MAX; unsigned indexmininnerp1 = 0;
        const unsigned pardetp1 = pardet[p1];
        //-For OPS
        const float3 normalvecp1 = normalvec[p1];
        float disminfsp1 = FLT_MAX, maxarccosp1 = 0; unsigned disminfsindexp1 = 0;
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVEMVCSOPS_StepIBox<tker,vem,vcs,ops>(false, p1, pini, pfin, poscell, velrhop, pressvcs, pardet, normalvec, CTE.massf, pscellp1, velrhop1, pardetp1, arp1, rhobarp1, aijp1, paijp1
                    , dismininnerp1, indexmininnerp1, normalvecp1, maxarccosp1, disminfsp1, disminfsindexp1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVEMVCSOPS_StepIBox<tker,vem,vcs,ops>(true, p1, pini, pfin, poscell, velrhop, pressvcs, pardet, normalvec, CTE.massf, pscellp1, velrhop1, pardetp1, arp1, rhobarp1, aijp1, paijp1
                    , dismininnerp1, indexmininnerp1, normalvecp1, maxarccosp1, disminfsp1, disminfsindexp1);
            }
        }
        //-VEM
        if(vem) pressvem[p1] = CTE.cs0 * CTE.cs0 * dt * arp1;//-P^{VEM} from divergence of Vel
        //-VCS
        aij[p1] = aijp1; paij[p1] = paijp1; rhobari[p1] = rhobarp1; dismininner[p1] = dismininnerp1; indexmininner[p1] = indexmininnerp1;
        //-OPS
        maxarccos[p1] = maxarccosp1;
        disminfs[p1] = disminfsp1; disminindexfs[p1] = disminfsindexp1;

        //if(pardetp1==2|| pardetp1 == 1) psig[p1]=0.;

    }

    if(vcs){
        if (p < n){
            //-VCS
            const unsigned p1 = p + pinit;      //-Number of particle.
            const float cs0 = CTE.cs0;
            const float rhobar0 = CTE.rhobarzero;
            const float rhopbysump1 = rhobari[p1];
            const double divcs = 1.0f / (rhobar0 * dt * dt * cs0 * cs0);
            const float eivcs = -(rhobar0 - rhopbysump1) / (rhobar0 * dt * dt);
            const float pressfin = (paij[p1] + eivcs) / (aij[p1] + divcs);
            pressvcsnewi[p1] = pressfin;
        }
    }

    
}

//-------------VCS Shift Vector--------------

template<TpKernel tker, bool vem, bool vcs, bool ops>
__device__ void KerRunVEMVCSOPS_StepIIBox(bool boundp2, unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop, const float* pressvcs, const float* pressvem, float massp2
    , const float4& pscellp1, const float4& velrhop1, const tmatrix3f& corrmatp1
    , float3& shiftvcsp1, const float& pressvcsp1, const float& pressvemp1, const double dt
    , const unsigned pardetp1, const float3 normalvecp1, float4& shiftposfsp1, float3& acevemp1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float fac = cufsph::GetKernel_Fac<tker>(rr2);
            const float frx = fac * drx, fry = fac * dry, frz = fac * drz; //-Gradients.
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            const float wab0 = cufsph::GetKernel_Wab<tker>(CTE.dp * CTE.dp);
            float4 velrhop2 = velrhop[p2];
            const float rhopp1 = velrhop1.w, rhopp2 = velrhop2.w;
            //const float pvcsp2 = pressvcs[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            const float vb = massp2 / rhopp2;


            if(vem){
                //-VEM
                if(!boundp2){
                    if (pressvemp1 <= 0 && pardetp1 == 3) {
                        const float prs = (pressvem[p2] - pressvemp1) / (rhopp1 * rhopp2);
                        const float p_vpm = -prs * massp2;
                        acevemp1.x += p_vpm * frx; acevemp1.y += p_vpm * fry; acevemp1.z += p_vpm * frz;
                    }
                    else {
                        const float prs = (pressvemp1 + pressvem[p2]) / (rhopp1 * rhopp2);
                        const float p_vpm = -prs * massp2;
                        acevemp1.x += p_vpm * frx; acevemp1.y += p_vpm * fry; acevemp1.z += p_vpm * frz;
                    }
                    //const float prs = (pressvemp1 + pressvem[p2]) / (rhopp1 * rhopp2);
                    //const float p_vpm = -prs * massp2;
                    //acevemp1.x += p_vpm * frx; acevemp1.y += p_vpm * fry; acevemp1.z += p_vpm * frz;

                }
            }

            if(vcs)
            {
                //-VCS
                double pconst = -dt * dt / rhopp1 * (pressvcs[p2] - pressvcsp1) * vb;//--------------------------
                //double pconst = -dt / rhopp1 * (pressvcs[p2] - pressvcsp1) * vb;
                shiftvcsp1.x += (pconst * (corrmatp1.a11 * frx + corrmatp1.a12 * fry + corrmatp1.a13 * frz));
                shiftvcsp1.y += (pconst * (corrmatp1.a21 * frx + corrmatp1.a22 * fry + corrmatp1.a23 * frz));
                shiftvcsp1.z += (pconst * (corrmatp1.a31 * frx + corrmatp1.a32 * fry + corrmatp1.a33 * frz));
            }
            
            if(ops){
            //-OPS
                const float wab_wab0 = wab / wab0;
                if (pardetp1 == 2 || pardetp1 == 3) {
                    shiftposfsp1.x += 1.0f * vb * frx * (0.2f * powf(wab_wab0, 4.0f));
                    shiftposfsp1.y += 1.0f * vb * fry * (0.2f * powf(wab_wab0, 4.0f));
                    shiftposfsp1.z += 1.0f * vb * frz * (0.2f * powf(wab_wab0, 4.0f));
                }
                else if (pardetp1 == 4) {
                    shiftposfsp1.x += 1.0f * vb * frx * (1.0f + 0.2f * powf(wab_wab0, 4.0f));
                    shiftposfsp1.y += 1.0f * vb * fry * (1.0f + 0.2f * powf(wab_wab0, 4.0f));
                    shiftposfsp1.z += 1.0f * vb * frz * (1.0f + 0.2f * powf(wab_wab0, 4.0f));
                }
            }
        }
    }
}


template<TpKernel tker, bool vem, bool vcs, bool ops>
__global__ void KerRunVEMVCSOPS_StepII(unsigned n, unsigned pinit
    , int scelldiv, int4 nc, int3 cellzero, const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, const unsigned* pardet, const float* pressvcs, const float* pressvem
    , float3 *acevem, const tmatrix3f* corrmat, const float3 *normalvec, float3* shiftvcs, float4* shiftposfs, const double dt
    , float* maxarccos, float* disminfs, unsigned* disminindexfs) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        const unsigned pardetp1 = pardet[p1];
        const tmatrix3f corrmatp1 = corrmat[p1];
        const float3 normalvecp1 = normalvec[p1];
        //-VEM
        const float pressvemp1 = vem? pressvem[p1]:FLT_MAX;
        float3 acevemp1 = make_float3(0, 0, 0);
        //-VCS
        float pressvcsp1 = vcs?pressvcs[p1]:FLT_MAX;
        float3 shiftvcsp1 = make_float3(0, 0, 0);
        //-OPS
        float4 shiftposfsp1 = make_float4(0, 0, 0, 0);


        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVEMVCSOPS_StepIIBox<tker,vem,vcs,ops>(false, p1, pini, pfin, poscell, velrhop, pressvcs, pressvem, CTE.massf, pscellp1, velrhop1, corrmatp1, shiftvcsp1, pressvcsp1, pressvemp1, dt, pardetp1, normalvecp1, shiftposfsp1, acevemp1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerRunVEMVCSOPS_StepIIBox<tker,vem,vcs,ops>(true, p1, pini, pfin, poscell, velrhop, pressvcs, pressvem, CTE.massf, pscellp1, velrhop1, corrmatp1, shiftvcsp1, pressvcsp1, pressvemp1, dt, pardetp1, normalvecp1, shiftposfsp1, acevemp1);
            }
        }

        if(vem)
        {
            //-VEM
            acevem[p1].x = acevemp1.x;
            acevem[p1].y = acevemp1.y;
            acevem[p1].z = acevemp1.z;
        }

        if(vcs)
        {
            //-VCS
            const float maxdist = CTE.kernelh / 20.0f;
            //const double shiftvcsnorm= sqrt(shiftvcsp1.x * shiftvcsp1.x + shiftvcsp1.y * shiftvcsp1.y + shiftvcsp1.z * shiftvcsp1.z);
            shiftvcsp1.x = (fabs(shiftvcsp1.x) < maxdist ? shiftvcsp1.x : (shiftvcsp1.x >= 0 ? maxdist : -maxdist));
            shiftvcsp1.y = (fabs(shiftvcsp1.y) < maxdist ? shiftvcsp1.y : (shiftvcsp1.y >= 0 ? maxdist : -maxdist));
            shiftvcsp1.z = (fabs(shiftvcsp1.z) < maxdist ? shiftvcsp1.z : (shiftvcsp1.z >= 0 ? maxdist : -maxdist));
            shiftvcs[p1] = (pardetp1 == 1) ? make_float3(0, 0, 0) : shiftvcsp1;
        
        }

        if(ops)
        {
            //-OPS
            if (pardetp1 == 2)//-free surface particles
            {
                unsigned sigma = maxarccos[p1] > PI / 6.0f ? 0 : 1;
                //float3 shiftvecaux = make_float(0);
                const float projectionlength = (normalvec[p1].x * shiftposfsp1.x + normalvec[p1].y * shiftposfsp1.y + normalvec[p1].z * shiftposfsp1.z);
                shiftposfsp1.x = sigma * (shiftposfsp1.x - projectionlength * normalvec[p1].x);
                shiftposfsp1.y = sigma * (shiftposfsp1.y - projectionlength * normalvec[p1].y);
                shiftposfsp1.z = sigma * (shiftposfsp1.z - projectionlength * normalvec[p1].z);
                //shiftposfs[p1] = shiftposfsp1;
            }
            else if (pardetp1 == 3)//-free surface vicinity particles
            {
                float phi = 0;
                if (disminfs[p1] != FLT_MAX) {
                    unsigned p2 = disminindexfs[p1];

                    const float4 pscellp2 = poscell[p2];
                    float rijx = -(pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w)));
                    float rijy = -(pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w)));
                    float rijz = -(pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w)));
                    float rij = sqrt(rijx * rijx + rijy * rijy + rijz * rijz);
                    phi = (rijx * normalvec[disminindexfs[p1]].x + rijy * normalvec[disminindexfs[p1]].y + rijz * normalvec[disminindexfs[p1]].z) / rij;
                    phi = phi > 0 ? phi : 0; // coefficient > 0 for free surface vicinity particles
                    shiftposfsp1.x = phi * shiftposfsp1.x; shiftposfsp1.y = phi * shiftposfsp1.y; shiftposfsp1.z = phi * shiftposfsp1.z;

                }
            }
            shiftposfs[p1] = shiftposfsp1;
        }


    }
}

//==============================================================================

template<TpKernel tker> void KerFreeSurfaceDetection(const StInterParmsg& t){
    dim3 sgridf = GetSimpleGridSize(t.fluidnum, t.bsfluid);
    const StDivDataGpu& dvd = t.divdatag;
    const int2* beginendcell = dvd.beginendcell;
    //-pardet- 0:wall particles 1:splash particles 2:free surface partiles 3:free-surface vicinity particles 4:inner particles 5:wall vicinity particles
    //-Parachute shape scan
    KerParachuteScan<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.simulate2d, t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
        , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.normalvec, t.pardet);
    //-Free surface vicinity particles
    KerFreeSurfaceVicinity<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.simulate2d, t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
        , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pardet);
    //-Wall vicinity particles
    KerWallVicinity<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.simulate2d, t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
        , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pardet);
}


template<TpKernel tker, bool vem, bool vcs, bool ops> void PreInteraction_gt3(const StInterParmsg& t){
    //    //-Collects kernel information.
    //#ifndef DISABLE_BSMODES
    //    if (t.kerinfo) {
    //        Interaction_ForcesT_KerInfo<tker, ftmode, lamsps, tdensity, shift, false>(t.kerinfo);
    //        return;
    //    }
    //#endif
    const StDivDataGpu& dvd = t.divdatag;
    const int2* beginendcell = dvd.beginendcell;

    //-Compute correction matrix for fluid
    if (t.fluidnum) {
        dim3 sgridf = GetSimpleGridSize(t.fluidnum, t.bsfluid);
        float* Ci; hipMalloc(&Ci, t.vnp * sizeof(float));//-
        //-Particle concentration, Corrective Matrix, Divergence of position vector
        KerGetAuxarray<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.simulate2d, t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
            , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.corrmat, t.pardet, t.veldiv,Ci);
        //-Normal vectors
        KerGetNormals<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.simulate2d, t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
            , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.corrmat, t.normalvec, Ci);
        hipFree(Ci);

        //-Particle type detection
        KerFreeSurfaceDetection<tker>(t);

        //-VEM-VCS-OPS
        if (vem || vcs || ops) {
            //-Auxiliary array
            float* Aij; float* PAij; float* Pressvcsnewi; float* Dismin; unsigned* Disminindex;
            hipMalloc(&Aij, t.vnp * sizeof(float));          hipMemset(Aij, 0, t.vnp * sizeof(float));
            hipMalloc(&PAij, t.vnp * sizeof(float));         hipMemset(PAij, 0, t.vnp * sizeof(float));
            hipMalloc(&Pressvcsnewi, t.vnp * sizeof(float)); hipMemset(Pressvcsnewi, 0, t.vnp * sizeof(float));
            hipMalloc(&Dismin, t.vnp * sizeof(float));       hipMemset(Dismin, 0, t.vnp * sizeof(float));
            hipMalloc(&Disminindex, t.vnp * sizeof(unsigned));  hipMemset(Disminindex, 0, t.vnp * sizeof(unsigned));
            float* MaxArcCos; float* Dismin2; unsigned* Disminindex2;
            hipMalloc(&MaxArcCos, t.vnp * sizeof(float));    hipMemset(MaxArcCos, 0, t.vnp * sizeof(float));
            hipMalloc(&Dismin2, t.vnp * sizeof(float));      hipMemset(Dismin2, 0, t.vnp * sizeof(float));
            hipMalloc(&Disminindex2, t.vnp * sizeof(unsigned)); hipMemset(Disminindex2, 0, t.vnp * sizeof(unsigned));

            //-VEM VCS OPS StepI: Compute -VEM: pressure, -VCS: pressure -OPS: max arccos, min distance of Free surface vicinity particles to free surface
            KerRunVEMVCSOPS_StepI<tker, vem, vcs, ops> << <sgridf, t.bsfluid, 0, t.stm >> > (t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
                , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pardet, t.pressvcs, t.pressvem, t.shiftposfs, t.normalvec
                , Aij, PAij, t.rhobar, Pressvcsnewi, Dismin, Disminindex, t.dt, MaxArcCos, Dismin2, Disminindex2, t.psi);

            if (vcs)
            {
                //-VCS Pressure of Free surface and vicinity particles
                KerRunVcsPress_I<tker> <<<sgridf, t.bsfluid, 0, t.stm >>> (t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
                , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pardet, t.pressvcs, Pressvcsnewi, Dismin, Disminindex);
                //-VCS pressure of boundary particles
                dim3 sgridb = GetSimpleGridSize(t.boundnum, t.bsbound);
                KerRunVcsPress_II<tker> <<<sgridb, t.bsbound, 0, t.stm >>> (t.boundnum, t.boundini, dvd.scelldiv, dvd.nc, dvd.cellzero
                , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pressvcs);

            }
            //-VEM VCS OPS Step II: Compute -VEM: acceleration, -VCS: shifting vector -OPS: shifting vector
            KerRunVEMVCSOPS_StepII<tker, vem, vcs, ops> << <sgridf, t.bsfluid, 0, t.stm >> > (t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
                , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.pardet, t.pressvcs, t.pressvem, t.acevem, t.corrmat
                , t.normalvec, t.shiftvcs, t.shiftposfs, t.dt, MaxArcCos, Dismin2, Disminindex2);

            hipFree(Aij); hipFree(PAij); hipFree(Pressvcsnewi); hipFree(Dismin); hipFree(Disminindex);
            hipFree(MaxArcCos); hipFree(Dismin2); hipFree(Disminindex2);
        }
    }
}

template<TpKernel tker, bool vem, bool vcs> void PreInteraction_gt2(const StInterParmsg& t) {
    if (t.tops)  PreInteraction_gt3<tker, vem, vcs, true>(t);
    else         PreInteraction_gt3<tker, vem, vcs, false>(t);
}
//==============================================================================
template<TpKernel tker, bool vem> void PreInteraction_gt1(const StInterParmsg& t) {
    if (t.tvol)  PreInteraction_gt2<tker, vem, true>(t);
    else         PreInteraction_gt2<tker, vem, false>(t);
}
//==============================================================================
template<TpKernel tker> void PreInteraction_gt0(const StInterParmsg& t) {
    if (t.tdiv)   PreInteraction_gt1<tker, true>(t);
    else PreInteraction_gt1<tker, false>(t);
}

//==============================================================================
void PreInteraction(const StInterParmsg& t) {
#ifdef FAST_COMPILATION
    if (t.tkernel != KERNEL_Wendland)throw "Extra kernels are disabled for FastCompilation...";
    PreInteraction_gt0<KERNEL_Wendland>(t);
#else
    if (t.tkernel == KERNEL_Wendland)     PreInteraction_gt0<KERNEL_Wendland>(t);
#ifndef DISABLE_KERNELS_EXTRA
    else if (t.tkernel == KERNEL_Cubic)   PreInteraction_gt0<KERNEL_Cubic   >(t);
#endif
#endif
}




//=============================Rho0 ref && N0 ref====================================
template<TpKernel tker>
__device__ void KerGetRhop1Np1Box(unsigned p1, const unsigned& pini, const unsigned& pfin
    , const float4* poscell, const float4* velrhop
    , float massp2, const float4& pscellp1, const float4& velrhop1, float& rhobarp1,unsigned &np2inp1p1)
{
    for (int p2 = pini; p2 < pfin; p2++) {
        const float4 pscellp2 = poscell[p2];
        float drx = pscellp1.x - pscellp2.x + CTE.poscellsize * (PSCEL_GetfX(pscellp1.w) - PSCEL_GetfX(pscellp2.w));
        float dry = pscellp1.y - pscellp2.y + CTE.poscellsize * (PSCEL_GetfY(pscellp1.w) - PSCEL_GetfY(pscellp2.w));
        float drz = pscellp1.z - pscellp2.z + CTE.poscellsize * (PSCEL_GetfZ(pscellp1.w) - PSCEL_GetfZ(pscellp2.w));
        //if (symm)dry = pscellp1.y + pscellp2.y + CTE.poscellsize * PSCEL_GetfY(pscellp2.w); //<vs_syymmetry>
        const float rr2 = drx * drx + dry * dry + drz * drz;
        if (rr2 <= CTE.kernelsize2 && rr2 >= ALMOSTZERO) {
            //-Computes kernel.
            const float wab = cufsph::GetKernel_Wab<tker>(rr2);
            float4 velrhop2 = velrhop[p2];
            //if (symm)velrhop2.y = -velrhop2.y; //<vs_syymmetry>
            //const float vb = massp2 / velrhop2.w;
            rhobarp1 += wab * massp2;
            np2inp1p1 +=1;
        }
    }
}

template<TpKernel tker>
__global__ void KerGetRhop1Np1(unsigned n, unsigned pinit, int scelldiv, int4 nc, int3 cellzero
    , const int2* begincell, unsigned cellfluid, const unsigned* dcell
    , const float4* poscell, const float4* velrhop, float *rhobar, unsigned *n0) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; //-Number of particle.
    if (p < n) {
        const unsigned p1 = p + pinit;      //-Number of particle.

        //-Obtains basic data of particle p1.
        const float4 pscellp1 = poscell[p1];
        const float4 velrhop1 = velrhop[p1];
        float rhobarp1 = cufsph::GetKernel_Wab<tker>(0) * CTE.massf;
        unsigned np2inp1p1= 0;
        //float cp1 = cufsph::GetKernel_Wab<tker>(0) * CTE.massf / velrhop1.w;//-add p1
        //-Obtains neighborhood search limits.
        int ini1, fin1, ini2, fin2, ini3, fin3;
        cunsearch::InitCte(dcell[p1], scelldiv, nc, cellzero, ini1, fin1, ini2, fin2, ini3, fin3);

        //-Interaction with fluids.
        ini3 += cellfluid; fin3 += cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetRhop1Np1Box<tker>(p1, pini, pfin, poscell, velrhop, CTE.massf, pscellp1, velrhop1, rhobarp1,np2inp1p1);
            }
        }
        //-Interaction with boundaries.
        ini3 -= cellfluid; fin3 -= cellfluid;
        for (int c3 = ini3; c3 < fin3; c3 += nc.w)for (int c2 = ini2; c2 < fin2; c2 += nc.x) {
            unsigned pini, pfin = 0;  cunsearch::ParticleRange(c2, c3, ini1, fin1, begincell, pini, pfin);
            if (pfin) {
                KerGetRhop1Np1Box<tker>(p1, pini, pfin, poscell, velrhop, CTE.massb, pscellp1, velrhop1, rhobarp1,np2inp1p1);
            }
        }
        rhobar[p1] = rhobarp1;
        n0[p1] = np2inp1p1;

        //printf("cip1 = {%d, %7.6f, %d}\n",
        //    p1, rhobar[p1], n0[p1]);
    }


}

template<TpKernel tker>
void ComputeValue_gt0(const StInterParmsg& t,float &rho0refg, unsigned &n0refg)
{
    const StDivDataGpu& dvd = t.divdatag;
    const int2* beginendcell = dvd.beginendcell;

    float *Rhobar; hipMalloc(&Rhobar, t.vnp * sizeof(float));hipMemset(Rhobar,0, t.vnp * sizeof(float));
    unsigned *Np2inp1; hipMalloc(&Np2inp1, t.vnp * sizeof(unsigned)); hipMemset(Np2inp1, 0, t.vnp * sizeof(unsigned));

    //-Reference Rho0 && N0 ref at beginning
    if (t.fluidnum) {
        dim3 sgridf = GetSimpleGridSize(t.fluidnum, t.bsfluid);
        KerGetRhop1Np1<tker> << <sgridf, t.bsfluid, 0, t.stm >> > (t.fluidnum, t.fluidini, dvd.scelldiv, dvd.nc, dvd.cellzero
            , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop,Rhobar,Np2inp1);
    }

    thrust::device_vector<float> rhobarvec(t.vnp);
    thrust::device_vector<unsigned> np2inp1vec(t.vnp);
    thrust::copy(Rhobar, Rhobar+ t.vnp, rhobarvec.begin());
    thrust::copy(Np2inp1, Np2inp1+ t.vnp, np2inp1vec.begin());

    thrust::device_vector<float>::iterator max_iter = thrust::max_element(rhobarvec.begin(), rhobarvec.end());
    int max_index = max_iter - rhobarvec.begin();
    float max_value = *max_iter;
    thrust::device_vector<unsigned>::iterator max_iter2 = thrust::max_element(np2inp1vec.begin(), np2inp1vec.end());
    int max_index2 = max_iter2 - np2inp1vec.begin();
    unsigned max_value2 = *max_iter2;
    hipFree(Rhobar);hipFree(Np2inp1);

    rho0refg = max_value;n0refg=max_value2;

    ////-Normal vector of boundary 
    if (t.boundnum)
    {
        dim3 sgridb = GetSimpleGridSize(t.vnpb, t.bsbound);
        float* Ci; hipMalloc(&Ci, t.vnp * sizeof(float));//-particle concentration �Ƿ���Ҫ��ǰ�����ڴ棬����̫��ʱ
        KerGetAuxarrayBound<tker> << <sgridb, t.bsbound, 0, t.stm >> > (t.simulate2d, t.vnpb, t.boundini, dvd.scelldiv, dvd.nc, dvd.cellzero
            , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.corrmat, t.pardet, Ci);
        KerGetNormalsBound<tker> << <sgridb, t.bsbound, 0, t.stm >> > (t.simulate2d, t.vnpb, t.boundini, dvd.scelldiv, dvd.nc, dvd.cellzero
            , dvd.beginendcell, dvd.cellfluid, t.dcell, t.poscell, t.velrhop, t.corrmat, t.normalvec, Ci);
        hipFree(Ci);
    }

}


void ComputeValue(const StInterParmsg& t, float& rho0refg, unsigned& n0refg) {
#ifdef FAST_COMPILATION
    if (t.tkernel != KERNEL_Wendland)throw "Extra kernels are disabled for FastCompilation...";
    ComputeValue_gt0<KERNEL_Wendland>(t);
#else
    if (t.tkernel == KERNEL_Wendland)     ComputeValue_gt0<KERNEL_Wendland>(t, rho0refg, n0refg);
#ifndef DISABLE_KERNELS_EXTRA
    else if (t.tkernel == KERNEL_Cubic)   ComputeValue_gt0<KERNEL_Cubic   >(t, rho0refg, n0refg);
#endif
#endif
}

//------------------------------------------------------------------------------
/// Returns the corrected position after applying periodic conditions.
/// Devuelve la posicion corregida tras aplicar condiciones periodicas.
//------------------------------------------------------------------------------
__device__ float4 KerComputePosCell(const double3 &ps,const double3 &mapposmin,float poscellsize)
{
  const double dx=ps.x-mapposmin.x;
  const double dy=ps.y-mapposmin.y;
  const double dz=ps.z-mapposmin.z;
  int cx=int(dx/poscellsize);
  int cy=int(dy/poscellsize);
  int cz=int(dz/poscellsize);
  cx=(cx>=0? cx: 0);
  cy=(cy>=0? cy: 0);
  cz=(cz>=0? cz: 0);
  const float px=float(dx-(double(poscellsize)*cx));
  const float py=float(dy-(double(poscellsize)*cy));
  const float pz=float(dz-(double(poscellsize)*cz));
  const float pw=__uint_as_float(PSCEL_Code(cx,cy,cz));
  return(make_float4(px,py,pz,pw));
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected bondary and fluid.
//------------------------------------------------------------------------------
template<TpKernel tker,bool sim2d,TpSlipMode tslip> __global__ void KerInteractionMdbcCorrection_Fast
  (unsigned n,unsigned nbound,float determlimit,float mdbcthreshold
  ,double3 mapposmin,float poscellsize,const float4 *poscell
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  const unsigned p1=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<n){
    const float3 bnormalp1=boundnormal[p1];
    if(bnormalp1.x!=0 || bnormalp1.y!=0 || bnormalp1.z!=0){
      float rhopfinal=FLT_MAX;
      float3 velrhopfinal=make_float3(0,0,0);
      float sumwab=0;

      //-Calculates ghost node position.
      double3 gposp1=make_double3(posxy[p1].x+bnormalp1.x,posxy[p1].y+bnormalp1.y,posz[p1]+bnormalp1.z);
      gposp1=(CTE.periactive!=0? KerUpdatePeriodicPos(gposp1): gposp1); //-Corrected interface Position.
      const float4 gpscellp1=KerComputePosCell(gposp1,mapposmin,poscellsize);

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);                              // -Only for velocity
      tmatrix3f a_corr2; if(sim2d) cumath::Tmatrix3fReset(a_corr2); //-Only for 2D.
      tmatrix4f a_corr3; if(!sim2d)cumath::Tmatrix4fReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(gposp1.x,gposp1.y,gposp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Boundary-Fluid interaction.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const float4 pscellp2=poscell[p2];
          float drx=gpscellp1.x-pscellp2.x + CTE.poscellsize*(PSCEL_GetfX(gpscellp1.w)-PSCEL_GetfX(pscellp2.w));
          float dry=gpscellp1.y-pscellp2.y + CTE.poscellsize*(PSCEL_GetfY(gpscellp1.w)-PSCEL_GetfY(pscellp2.w));
          float drz=gpscellp1.z-pscellp2.z + CTE.poscellsize*(PSCEL_GetfZ(gpscellp1.w)-PSCEL_GetfZ(pscellp2.w));
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            //===== Get mass and volume of particle p2 =====
            const float4 velrhopp2=velrhop[p2];
            float massp2=CTE.massf;
            const float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            sumwab+=vwab;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity =====
            if(tslip!=SLIP_Vel0) {
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      if(sumwab>=mdbcthreshold || (mdbcthreshold>=2 && sumwab+2>=mdbcthreshold)){
        const float3 dpos=make_float3(-bnormalp1.x,-bnormalp1.y,-bnormalp1.z); //-Boundary particle position - ghost node position.
        if(sim2d){
          const double determ=cumath::Determinant3x3dbl(a_corr2);
          if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
            const tmatrix3f invacorr2=cumath::InverseMatrix3x3dbl(a_corr2,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr2.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
        else{
          const double determ=cumath::Determinant4x4dbl(a_corr3);
          if(fabs(determ)>=determlimit){
            const tmatrix4f invacorr3=cumath::InverseMatrix4x4dbl(a_corr3,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr3.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
        //-Store the results.
        rhopfinal=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
        if(tslip==SLIP_Vel0){//-DBC vel=0
          velrhop[p1].w=rhopfinal;
        }
        if(tslip==SLIP_NoSlip){//-No-Slip
          const float3 v=motionvel[p1];
          velrhop[p1]=make_float4(v.x+v.x-velrhopfinal.x,v.y+v.y-velrhopfinal.y,v.z+v.z-velrhopfinal.z,rhopfinal);
        }
        if(tslip==SLIP_FreeSlip){//-No-Penetration and free slip    SHABA
          float3 FSVelFinal; // final free slip boundary velocity
          const float3 v=motionvel[p1];
          float motion=sqrt(v.x*v.x+v.y*v.y+v.z*v.z); // to check if boundary moving
          float norm=sqrt(bnormalp1.x*bnormalp1.x+bnormalp1.y*bnormalp1.y+bnormalp1.z*bnormalp1.z);
          float3 normal; // creating a normailsed boundary normal
          normal.x=fabs(bnormalp1.x)/norm; normal.y=fabs(bnormalp1.y)/norm; normal.z=fabs(bnormalp1.z)/norm;

          // finding the velocity componants normal and tangential to boundary 
          float3 normvel=make_float3(velrhopfinal.x*normal.x,velrhopfinal.y*normal.y,velrhopfinal.z*normal.z); // velocity in direction of normal pointin ginto fluid)
          float3 tangvel=make_float3(velrhopfinal.x-normvel.x,velrhopfinal.y-normvel.y,velrhopfinal.z-normvel.z); // velocity tangential to normal

          if (motion > 0) { // if moving boundary
              float3 normmot=make_float3(v.x*normal.x,v.y*normal.y,v.z*normal.z); // boundary motion in direction normal to boundary 
              FSVelFinal=make_float3(normmot.x+normmot.x-normvel.x,normmot.y+normmot.y-normvel.y,normmot.z+normmot.z-normvel.z);
              // only velocity in normal direction for no-penetration
              // fluid sees zero velocity in the tangetial direction
          }
          else {
              FSVelFinal=make_float3(tangvel.x-normvel.x,tangvel.y-normvel.y,tangvel.z-normvel.z);
              // tangential velocity equal to fluid velocity for free slip
              // normal velocity reversed for no-penetration
          }

          // Save the velocity and density
          velrhop[p1]=make_float4(FSVelFinal.x,FSVelFinal.y,FSVelFinal.z,rhopfinal);
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected bondary and fluid.
//------------------------------------------------------------------------------
template<TpKernel tker,bool sim2d,TpSlipMode tslip> __global__ void KerInteractionMdbcCorrection_Dbl
  (unsigned n,unsigned nbound,float determlimit,float mdbcthreshold
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  const unsigned p1=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<n){
    const float3 bnormalp1=boundnormal[p1];
    if(bnormalp1.x!=0 || bnormalp1.y!=0 || bnormalp1.z!=0){
      float rhopfinal=FLT_MAX;
      float3 velrhopfinal=make_float3(0,0,0);
      float sumwab=0;

      //-Calculates ghost node position.
      double3 gposp1=make_double3(posxy[p1].x+bnormalp1.x,posxy[p1].y+bnormalp1.y,posz[p1]+bnormalp1.z);
      gposp1=(CTE.periactive!=0? KerUpdatePeriodicPos(gposp1): gposp1); //-Corrected interface Position.
      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);                              // -Only for velocity
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(gposp1.x,gposp1.y,gposp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Boundary-Fluid interaction.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double2 p2xy=posxy[p2];
          const float drx=float(gposp1.x-p2xy.x);
          const float dry=float(gposp1.y-p2xy.y);
          const float drz=float(gposp1.z-posz[p2]);
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            //===== Get mass and volume of particle p2 =====
            const float4 velrhopp2=velrhop[p2];
            float massp2=CTE.massf;
            const float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            sumwab+=vwab;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity =====
            if(tslip!=SLIP_Vel0) {
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      if(sumwab>=mdbcthreshold){
        const float3 dpos=make_float3(-bnormalp1.x,-bnormalp1.y,-bnormalp1.z); //-Boundary particle position - ghost node position.
        if(sim2d){
          const double determ=cumath::Determinant3x3(a_corr2);
          if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
            const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr2.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
        else{
          const double determ=cumath::Determinant4x4(a_corr3);
          if(fabs(determ)>=determlimit){
            const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr3.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
        //-Store the results.
        rhopfinal=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
        if(tslip==SLIP_Vel0){//-DBC vel=0
          velrhop[p1].w=rhopfinal;
        }
        if(tslip==SLIP_NoSlip){//-No-Slip
          const float3 v=motionvel[p1];
          velrhop[p1]=make_float4(v.x+v.x-velrhopfinal.x,v.y+v.y-velrhopfinal.y,v.z+v.z-velrhopfinal.z,rhopfinal);
        }
        if(tslip==SLIP_FreeSlip){//-No-Penetration and free slip    SHABA
          float3 FSVelFinal; // final free slip boundary velocity
          const float3 v=motionvel[p1];
          float motion=sqrt(v.x*v.x+v.y*v.y+v.z*v.z); // to check if boundary moving
          float norm=sqrt(bnormalp1.x*bnormalp1.x+bnormalp1.y*bnormalp1.y+bnormalp1.z*bnormalp1.z);
          float3 normal; // creating a normailsed boundary normal
          normal.x=fabs(bnormalp1.x)/norm; normal.y=fabs(bnormalp1.y)/norm; normal.z=fabs(bnormalp1.z)/norm;

          // finding the velocity componants normal and tangential to boundary 
          float3 normvel=make_float3(velrhopfinal.x*normal.x,velrhopfinal.y*normal.y,velrhopfinal.z*normal.z); // velocity in direction of normal pointin ginto fluid)
          float3 tangvel=make_float3(velrhopfinal.x-normvel.x,velrhopfinal.y-normvel.y,velrhopfinal.z-normvel.z); // velocity tangential to normal

          if (motion > 0) { // if moving boundary
              float3 normmot=make_float3(v.x*normal.x,v.y*normal.y,v.z*normal.z); // boundary motion in direction normal to boundary 
              FSVelFinal=make_float3(normmot.x+normmot.x-normvel.x,normmot.y+normmot.y-normvel.y,normmot.z+normmot.z-normvel.z);
              // only velocity in normal direction for no-penetration
              // fluid sees zero velocity in the tangetial direction
          }
          else {
              FSVelFinal=make_float3(tangvel.x-normvel.x,tangvel.y-normvel.y,tangvel.z-normvel.z);
              // tangential velocity equal to fluid velocity for free slip
              // normal velocity reversed for no-penetration
          }

          // Save the velocity and density
          velrhop[p1]=make_float4(FSVelFinal.x,FSVelFinal.y,FSVelFinal.z,rhopfinal);
        }
      }
    }
  }
}


//==============================================================================
/// Calculates extrapolated data on boundary particles from fluid domain for mDBC.
/// Calcula datos extrapolados en el contorno para mDBC.
//==============================================================================
template<TpKernel tker,bool sim2d,TpSlipMode tslip> void Interaction_MdbcCorrectionT2(
  bool fastsingle,unsigned n,unsigned nbound,float mdbcthreshold,const StDivDataGpu &dvd
  ,const tdouble3 &mapposmin,const double2 *posxy,const double *posz,const float4 *poscell
  ,const typecode *code,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel
  ,float4 *velrhop)
{
  const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
  const float determlimit=1e-3f;
  //-Interaction GhostBoundaryNodes-Fluid.
  if(n){
    const unsigned bsbound=128;
    dim3 sgridb=cusph::GetSimpleGridSize(n,bsbound);
    if(fastsingle){//-mDBC-Fast_v2
      KerInteractionMdbcCorrection_Fast <tker,sim2d,tslip> <<<sgridb,bsbound>>> (n,nbound
        ,determlimit,mdbcthreshold,Double3(mapposmin),dvd.poscellsize,poscell
        ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,posxy,posz,code,idp,boundnormal,motionvel,velrhop);
    }
    else{//-mDBC_v0
      KerInteractionMdbcCorrection_Dbl <tker,sim2d,tslip> <<<sgridb,bsbound>>> (n,nbound
        ,determlimit,mdbcthreshold,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,posxy,posz,code,idp,boundnormal,motionvel,velrhop);
    }
  }
}
//==============================================================================
template<TpKernel tker> void Interaction_MdbcCorrectionT(bool simulate2d
  ,TpSlipMode slipmode,bool fastsingle,unsigned n,unsigned nbound
  ,float mdbcthreshold,const StDivDataGpu &dvd,const tdouble3 &mapposmin
  ,const double2 *posxy,const double *posz,const float4 *poscell,const typecode *code
  ,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  switch(slipmode){
    case SLIP_Vel0:{ const TpSlipMode tslip=SLIP_Vel0;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#ifndef DISABLE_MDBC_EXTRAMODES
    case SLIP_NoSlip:{ const TpSlipMode tslip=SLIP_NoSlip;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
    case SLIP_FreeSlip:{ const TpSlipMode tslip=SLIP_FreeSlip;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#endif
    default: throw "SlipMode unknown at Interaction_MdbcCorrectionT().";
  }
}
//==============================================================================
/// Calculates extrapolated data on boundary particles from fluid domain for mDBC.
/// Calcula datos extrapolados en el contorno para mDBC.
//==============================================================================
void Interaction_MdbcCorrection(TpKernel tkernel,bool simulate2d,TpSlipMode slipmode
  ,bool fastsingle,unsigned n,unsigned nbound,float mdbcthreshold
  ,const StDivDataGpu &dvd,const tdouble3 &mapposmin
  ,const double2 *posxy,const double *posz,const float4 *poscell,const typecode *code
  ,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  switch(tkernel){
    case KERNEL_Wendland:{ const TpKernel tker=KERNEL_Wendland;
      Interaction_MdbcCorrectionT <tker> (simulate2d,slipmode,fastsingle,n,nbound,mdbcthreshold
        ,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#ifndef DISABLE_KERNELS_EXTRA
    case KERNEL_Cubic:{ const TpKernel tker=KERNEL_Cubic;
      Interaction_MdbcCorrectionT <tker> (simulate2d,slipmode,fastsingle,n,nbound,mdbcthreshold
        ,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#endif
    default: throw "Kernel unknown at Interaction_MdbcCorrection().";
  }
}


//##############################################################################
//# Kernels for DEM interaction.
//# Kernels para interaccion DEM.
//##############################################################################
//------------------------------------------------------------------------------
/// DEM interaction of a particle with a set of particles. (Float-Float/Bound)
/// Realiza la interaccion DEM de una particula con un conjunto de ellas. (Float-Float/Bound)
//------------------------------------------------------------------------------
__device__ void KerInteractionForcesDemBox 
  (bool boundp2,const unsigned &pini,const unsigned &pfin
  ,const float4 *demdata,float dtforce
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float4 &pscellp1,const float4 &velp1,typecode tavp1,float masstotp1
  ,float ftmassp1,float taup1,float kfricp1,float restitup1
  ,float3 &acep1,float &demdtp1)
{
  for(int p2=pini;p2<pfin;p2++){
    const typecode codep2=code[p2];
    if(CODE_IsNotFluid(codep2) && tavp1!=CODE_GetTypeAndValue(codep2)){
      const float4 pscellp2=poscell[p2];
      const float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(PSCEL_GetfX(pscellp1.w)-PSCEL_GetfX(pscellp2.w));
      const float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(PSCEL_GetfY(pscellp1.w)-PSCEL_GetfY(pscellp2.w));
      const float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(PSCEL_GetfZ(pscellp1.w)-PSCEL_GetfZ(pscellp2.w));
      const float rr2=drx*drx+dry*dry+drz*drz;
      const float rad=sqrt(rr2);

      //-Computes maximum value of demdt.
      float4 demdatap2=demdata[CODE_GetTypeAndValue(codep2)];
      const float nu_mass=(boundp2? masstotp1/2: masstotp1*demdatap2.x/(masstotp1+demdatap2.x)); //-With boundary takes the actual mass of floating 1. | Con boundary toma la propia masa del floating 1.
      const float kn=4/(3*(taup1+demdatap2.y))*sqrt(CTE.dp/4); //-Generalized rigidity - Lemieux 2008.
      const float dvx=velp1.x-velrhop[p2].x, dvy=velp1.y-velrhop[p2].y, dvz=velp1.z-velrhop[p2].z; //vji
      const float nx=drx/rad, ny=dry/rad, nz=drz/rad; //-normal_ji             
      const float vn=dvx*nx+dvy*ny+dvz*nz; //-vji.nji    
      const float demvisc=0.2f/(3.21f*(pow(nu_mass/kn,0.4f)*pow(fabs(vn),-0.2f))/40.f);
      if(demdtp1<demvisc)demdtp1=demvisc;

      const float over_lap=1.0f*CTE.dp-rad; //-(ri+rj)-|dij|
      if(over_lap>0.0f){ //-Contact.
        //-Normal.
        const float eij=(restitup1+demdatap2.w)/2;
        const float gn=-(2.0f*log(eij)*sqrt(nu_mass*kn))/(sqrt(float(PI)+log(eij)*log(eij))); //-Generalized damping - Cummins 2010.
        //const float gn=0.08f*sqrt(nu_mass*sqrt(CTE.dp/2)/((taup1+demdatap2.y)/2)); //-generalized damping - Lemieux 2008.
        const float rep=kn*pow(over_lap,1.5f);
        const float fn=rep-gn*pow(over_lap,0.25f)*vn;
        float acef=fn/ftmassp1; //-Divides by the mass of particle to obtain the acceleration.
        acep1.x+=(acef*nx); acep1.y+=(acef*ny); acep1.z+=(acef*nz); //-Force is applied in the normal between the particles.
        //-Tangencial.
        const float dvxt=dvx-vn*nx, dvyt=dvy-vn*ny, dvzt=dvz-vn*nz; //Vji_t
        const float vt=sqrt(dvxt*dvxt + dvyt*dvyt + dvzt*dvzt);
        const float tx=(vt!=0? dvxt/vt: 0), ty=(vt!=0? dvyt/vt: 0), tz=(vt!=0? dvzt/vt: 0); //-Tang vel unit vector.
        const float ft_elast=2*(kn*dtforce-gn)*vt/7; //-Elastic frictional string -->  ft_elast=2*(kn*fdispl-gn*vt)/7; fdispl=dtforce*vt;
        const float kfric_ij=(kfricp1+demdatap2.z)/2;
        float ft=kfric_ij*fn*tanh(8*vt);  //-Coulomb.
        ft=(ft<ft_elast? ft: ft_elast);   //-Not above yield criteria, visco-elastic model.
        acef=ft/ftmassp1; //-Divides by the mass of particle to obtain the acceleration.
        acep1.x+=(acef*tx); acep1.y+=(acef*ty); acep1.z+=(acef*tz);
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
__global__ void KerInteractionForcesDem(unsigned nfloat
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,const float *ftomassp,float dtforce
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float3 *ace)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<nfloat){
    const unsigned p1=ftridp[p]; //-Number of particle.
    if(p1!=UINT_MAX){
      float demdtp1=0;
      float3 acep1=make_float3(0,0,0);

      //-Obtains basic data of particle p1.
      const float4 pscellp1=poscell[p1];
      const float4 velp1=velrhop[p1];
      const typecode cod=code[p1];
      const typecode tavp1=CODE_GetTypeAndValue(cod);
      const float4 rdata=demdata[tavp1];
      const float masstotp1=rdata.x;
      const float taup1=rdata.y;
      const float kfricp1=rdata.z;
      const float restitup1=rdata.w;
      const float ftmassp1=ftomassp[CODE_GetTypeValue(cod)];

      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Interaction with boundaries.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin)KerInteractionForcesDemBox (true ,pini,pfin,demdata,dtforce,poscell,velrhop,code,idp,pscellp1,velp1,tavp1,masstotp1,ftmassp1,taup1,kfricp1,restitup1,acep1,demdtp1);
      }

      //-Interaction with fluids.
      ini3+=cellfluid; fin3+=cellfluid;
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin)KerInteractionForcesDemBox (false,pini,pfin,demdata,dtforce,poscell,velrhop,code,idp,pscellp1,velp1,tavp1,masstotp1,ftmassp1,taup1,kfricp1,restitup1,acep1,demdtp1);
      }

      //-Stores results.
      if(acep1.x || acep1.y || acep1.z || demdtp1){
        float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
        if(viscdt[p1]<demdtp1)viscdt[p1]=demdtp1;
      }
    }
  }
}

#ifndef DISABLE_BSMODES
//==============================================================================
/// Collects kernel information.
//==============================================================================
void Interaction_ForcesDemT_KerInfo(StKerInfo *kerinfo)
{
#if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,int,int4,int3,const int2*,unsigned,const unsigned*,const unsigned*,const float4*,const float*,float,const float4*,const float4*,const typecode*,const unsigned*,float*,float3*);
    fun_ptr ptr=&KerInteractionForcesDem;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesdem_bs=qblocksize;
    kerinfo->forcesdem_rg=attr.numRegs;
    kerinfo->forcesdem_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesDem  blocksize:%u (%u)\n",qblocksize,0);
  }
  fcuda::Check_CudaErroorFun("Error collecting kernel information.");
#endif
}
#endif

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
void Interaction_ForcesDem(unsigned bsize,unsigned nfloat
  ,const StDivDataGpu &dvd,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,const float *ftomassp,float dtforce
  ,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,const unsigned *idp,float *viscdt,float3 *ace,StKerInfo *kerinfo)
{
  const int2* beginendcell=dvd.beginendcell;
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(kerinfo){
    Interaction_ForcesDemT_KerInfo(kerinfo);
    return;
  }
#endif
  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(nfloat){
    dim3 sgrid=GetSimpleGridSize(nfloat,bsize);
    KerInteractionForcesDem <<<sgrid,bsize>>> (nfloat
      ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcell,dvd.cellfluid,dcell
      ,ftridp,demdata,ftomassp,dtforce,poscell,velrhop,code,idp,viscdt,ace);
  }
}


//##############################################################################
//# Kernels for Laminar+SPS.
//##############################################################################
//------------------------------------------------------------------------------
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//------------------------------------------------------------------------------
__global__ void KerComputeSpsTau(unsigned n,unsigned pini,float smag,float blin
  ,const float4 *velrhop,const float2 *gradvelff,float2 *tauff)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; 
  if(p<n){
    const unsigned p1=p+pini;
    float2 rr=gradvelff[p1*3];   const float grad_xx=rr.x,grad_xy=rr.y;
           rr=gradvelff[p1*3+1]; const float grad_xz=rr.x,grad_yy=rr.y;
           rr=gradvelff[p1*3+2]; const float grad_yz=rr.x,grad_zz=rr.y;
    const float pow1=grad_xx*grad_xx + grad_yy*grad_yy + grad_zz*grad_zz;
    const float prr= grad_xy*grad_xy + grad_xz*grad_xz + grad_yz*grad_yz + pow1+pow1;
    const float visc_sps=smag*sqrt(prr);
    const float div_u=grad_xx+grad_yy+grad_zz;
    const float sps_k=(2.0f/3.0f)*visc_sps*div_u;
    const float sps_blin=blin*prr;
    const float sumsps=-(sps_k+sps_blin);
    const float twovisc_sps=(visc_sps+visc_sps);
    float one_rho2=1.0f/velrhop[p1].w;
    //-Computes new values of tau[].
    const float tau_xx=one_rho2*(twovisc_sps*grad_xx +sumsps);
    const float tau_xy=one_rho2*(visc_sps   *grad_xy);
    tauff[p1*3]=make_float2(tau_xx,tau_xy);
    const float tau_xz=one_rho2*(visc_sps   *grad_xz);
    const float tau_yy=one_rho2*(twovisc_sps*grad_yy +sumsps);
    tauff[p1*3+1]=make_float2(tau_xz,tau_yy);
    const float tau_yz=one_rho2*(visc_sps   *grad_yz);
    const float tau_zz=one_rho2*(twovisc_sps*grad_zz +sumsps);
    tauff[p1*3+2]=make_float2(tau_yz,tau_zz);
  }
}

//==============================================================================
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//==============================================================================
void ComputeSpsTau(unsigned np,unsigned npb,float smag,float blin
  ,const float4 *velrhop,const tsymatrix3f *gradvelg,tsymatrix3f *tau,hipStream_t stm)
{
  const unsigned npf=np-npb;
  if(npf){
    dim3 sgridf=GetSimpleGridSize(npf,SPHBSIZE);
    KerComputeSpsTau <<<sgridf,SPHBSIZE,0,stm>>> (npf,npb,smag,blin,velrhop,(const float2*)gradvelg,(float2*)tau);
  }
}


//##############################################################################
//# Kernels for Delta-SPH.
//# Kernels para Delta-SPH.
//##############################################################################
//------------------------------------------------------------------------------
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Anhade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//------------------------------------------------------------------------------
__global__ void KerAddDelta(unsigned n,const float *delta,float *ar)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    float rdelta=delta[p];
    if(rdelta!=FLT_MAX)ar[p]+=rdelta;
  }
}

//==============================================================================
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Anhade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//==============================================================================
void AddDelta(unsigned n,const float *delta,float *ar,hipStream_t stm){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerAddDelta <<<sgrid,SPHBSIZE,0,stm>>> (n,delta,ar);
  }
}


//##############################################################################
//# Kernels para ComputeStep (position)
//# Kernels for ComputeStep (position)
//##############################################################################
//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floatings> __global__ void KerComputeStepPos(unsigned n,unsigned pini
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floatings || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal && fluid){ //-Does not apply to periodic or floating particles. | No se aplica a particulas periodicas o floating.
      const double2 rmovxy=movxy[p];
      KerUpdatePos<periactive>(posxy[p],posz[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
    }
    //-In case of floating maintains the original position.
    //-En caso de floating mantiene la posicion original.
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos(byte periactive,bool floatings,unsigned np,unsigned npb
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetSimpleGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floatings)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floatings)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
  }
}

//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floatings> __global__ void KerComputeStepPos2(unsigned n,unsigned pini
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floatings || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal){//-Does not apply to periodic particles. | No se aplica a particulas periodicas
      if(fluid){//-Only applied for fluid displacement. | Solo se aplica desplazamiento al fluido.
        const double2 rmovxy=movxy[p];
        KerUpdatePos<periactive>(posxypre[p],poszpre[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
      }
      else{ //-Copy position of floating particles.
        posxy[p]=posxypre[p];
        posz[p]=poszpre[p];
      }
    }
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos2(byte periactive,bool floatings,unsigned np,unsigned npb
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetSimpleGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floatings)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floatings)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
  }
}



//##############################################################################
//# Kernels for motion.
//# Kernels para Motion
//##############################################################################
//------------------------------------------------------------------------------
/// Computes for a range of particles, their position according to idp[].
/// Calcula para un rango de particulas calcula su posicion segun idp[].
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    unsigned id=idp[p];
    if(idini<=id && id<idfin){
      if(CODE_IsNormal(code[p]))ridp[id-idini]=p;
    }
  }
}
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    const unsigned id=idp[p];
    if(idini<=id && id<idfin)ridp[id-idini]=p;
  }
}

//==============================================================================
/// Calculate particle position according to idp[]. When it does not find UINT_MAX.
/// When periactive is false it means there are no duplicate particles (periodic)
/// and all are CODE_NORMAL.
///
/// Calcula posicion de particulas segun idp[]. Cuando no la encuentra es UINT_MAX.
/// Cuando periactive es False sumpone que no hay particulas duplicadas (periodicas)
/// y todas son CODE_NORMAL.
//==============================================================================
void CalcRidp(bool periactive,unsigned np,unsigned pini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp){
  //-Assigns values UINT_MAX
  const unsigned nsel=idfin-idini;
  hipMemset(ridp,255,sizeof(unsigned)*nsel); 
  //-Computes position according to id. | Calcula posicion segun id.
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,code,idp,ridp);
    else          KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,idp,ridp);
  }
}

//------------------------------------------------------------------------------
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerMoveLinBound(unsigned n,unsigned ini,double3 mvpos,float3 mvvel
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      //-Computes displacement and updates position.
      KerUpdatePos<periactive>(posxy[pid],posz[pid],mvpos.x,mvpos.y,mvpos.z,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(mvvel.x,mvvel.y,mvvel.z,velrhop[pid].w);
    }
  }
}

//==============================================================================
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//==============================================================================
void MoveLinBound(byte periactive,unsigned np,unsigned ini,tdouble3 mvpos,tfloat3 mvvel
  ,const unsigned *ridp,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(periactive)KerMoveLinBound<true>  <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
  else          KerMoveLinBound<false> <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
}



//------------------------------------------------------------------------------
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive,bool simulate2d> __global__ void KerMoveMatBound(unsigned n,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code,float3 *boundnormal,float3 *normalvec)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      double2 rxy=posxy[pid];
      double3 rpos=make_double3(rxy.x,rxy.y,posz[pid]);
      //-Computes new position.
      double3 rpos2;
      rpos2.x= rpos.x*m.a11 + rpos.y*m.a12 + rpos.z*m.a13 + m.a14;
      rpos2.y= rpos.x*m.a21 + rpos.y*m.a22 + rpos.z*m.a23 + m.a24;
      rpos2.z= rpos.x*m.a31 + rpos.y*m.a32 + rpos.z*m.a33 + m.a34;
      if(simulate2d)rpos2.y=rpos.y;
      //-Computes displacement and updates position.
      const double dx=rpos2.x-rpos.x;
      const double dy=rpos2.y-rpos.y;
      const double dz=rpos2.z-rpos.z;
      KerUpdatePos<periactive>(make_double2(rpos.x,rpos.y),rpos.z,dx,dy,dz,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(float(dx/dt),float(dy/dt),float(dz/dt),velrhop[pid].w);
      //-Computes normal.
      if(boundnormal){
        const float3 bnor=boundnormal[pid];
        const double3 gs=make_double3(rpos.x+bnor.x,rpos.y+bnor.y,rpos.z+bnor.z);
        const double gs2x=gs.x*m.a11 + gs.y*m.a12 + gs.z*m.a13 + m.a14;
        const double gs2y=gs.x*m.a21 + gs.y*m.a22 + gs.z*m.a23 + m.a24;
        const double gs2z=gs.x*m.a31 + gs.y*m.a32 + gs.z*m.a33 + m.a34;
        boundnormal[pid]=make_float3(gs2x-rpos2.x,gs2y-rpos2.y,gs2z-rpos2.z);
      }
      if (normalvec) {
        const float3 bnor = normalvec[pid];
        const double3 gs = make_double3(rpos.x + bnor.x, rpos.y + bnor.y, rpos.z + bnor.z);
        const double gs2x = gs.x * m.a11 + gs.y * m.a12 + gs.z * m.a13 + m.a14;
        const double gs2y = gs.x * m.a21 + gs.y * m.a22 + gs.z * m.a23 + m.a24;
        const double gs2z = gs.x * m.a31 + gs.y * m.a32 + gs.z * m.a33 + m.a34;
        normalvec[pid] = make_float3(gs2x - rpos2.x, gs2y - rpos2.y, gs2z - rpos2.z);
      }
    }
  }
}

//==============================================================================
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//==============================================================================
void MoveMatBound(byte periactive,bool simulate2d,unsigned np,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code,float3 *boundnormal, float3 *normalvec)
{
  dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(periactive){ const bool peri=true;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal,normalvec);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal,normalvec);
  }
  else{ const bool peri=false;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal,normalvec);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal,normalvec);
  }
}

//------------------------------------------------------------------------------
/// Copy motion velocity to MotionVel[].
/// Copia velocidad de movimiento a MotionVel[].
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerCopyMotionVel(unsigned n
  ,const unsigned *ridpmv,const float4 *velrhop,float3 *motionvel)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p];
    if(pid>=0){
      //-Computes velocity.
      const float4 v=velrhop[pid];
      motionvel[pid]=make_float3(v.x,v.y,v.z);
    }
  }
}

//==============================================================================
/// Copy motion velocity to MotionVel[].
/// Copia velocidad de movimiento a MotionVel[].
//==============================================================================
void CopyMotionVel(unsigned nmoving,const unsigned *ridp,const float4 *velrhop,float3 *motionvel)
{
  dim3 sgrid=GetSimpleGridSize(nmoving,SPHBSIZE);
  KerCopyMotionVel<true>  <<<sgrid,SPHBSIZE>>> (nmoving,ridp,velrhop,motionvel);
}


//------------------------------------------------------------------------------
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//------------------------------------------------------------------------------
__global__ void KerFtNormalsUpdate(unsigned n,unsigned fpini
  ,double a11,double a12,double a13,double a21,double a22,double a23,double a31,double a32,double a33
  ,const unsigned *ftridp,float3 *boundnormal)
{
  const unsigned fp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of floating particle.
  if(fp<n){
    const unsigned p=ftridp[fp+fpini];
    if(p!=UINT_MAX){
      float3 rnor=boundnormal[p];
      const double nx=rnor.x;
      const double ny=rnor.y;
      const double nz=rnor.z;
      rnor.x=float(a11*nx + a12*ny + a13*nz);
      rnor.y=float(a21*nx + a22*ny + a23*nz);
      rnor.z=float(a31*nx + a32*ny + a33*nz);
      boundnormal[p]=rnor;
    }
  }
}

//==============================================================================
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//==============================================================================
void FtNormalsUpdate(unsigned np,unsigned ini,tmatrix4d m,const unsigned *ftridp
  ,float3 *boundnormal)
{
  dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(np)KerFtNormalsUpdate <<<sgrid,SPHBSIZE>>> (np,ini,m.a11,m.a12,m.a13
    ,m.a21,m.a22,m.a23,m.a31,m.a32,m.a33,ftridp,boundnormal);
}



//##############################################################################
//# Kernels for MLPistons motion.
//##############################################################################
//------------------------------------------------------------------------------
/// Applies movement and velocity of piston 1D to a group of particles.
/// Aplica movimiento y velocidad de piston 1D a conjunto de particulas.
//------------------------------------------------------------------------------
template<byte periactive> __global__ void KerMovePiston1d(unsigned n,unsigned idini
  ,double dp,double poszmin,unsigned poszcount,const byte *pistonid,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(p<n){
    const unsigned id=p+idini;
    int pid=ridpmv[id];
    if(pid>=0){
      const unsigned pisid=pistonid[CODE_GetTypeValue(code[pid])];
      if(pisid<255){
        const double2 rpxy=posxy[pid];
        const double rpz=posz[pid];
        const unsigned cz=unsigned((rpz-poszmin)/dp);
        const double rmovx=(cz<poszcount? movx[pisid*poszcount+cz]: 0);
        const float rvelx=float(cz<poszcount? velx[pisid*poszcount+cz]: 0);
        //-Updates position.
        KerUpdatePos<periactive>(rpxy,rpz,rmovx,0,0,false,pid,posxy,posz,dcell,code);
        //-Updates velocity.
        velrhop[pid].x=rvelx;
      }
    }
  }
}

//==============================================================================
/// Applies movement and velocity of piston 1D to a group of particles.
/// Aplica movimiento y velocidad de piston 1D a conjunto de particulas.
//==============================================================================
void MovePiston1d(bool periactive,unsigned np,unsigned idini
  ,double dp,double poszmin,unsigned poszcount,const byte *pistonid,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerMovePiston1d<true>  <<<sgrid,SPHBSIZE>>> (np,idini,dp,poszmin,poszcount,pistonid,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMovePiston1d<false> <<<sgrid,SPHBSIZE>>> (np,idini,dp,poszmin,poszcount,pistonid,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
  }
}

//------------------------------------------------------------------------------
/// Applies movement and velocity of piston 2D to a group of particles.
/// Aplica movimiento y velocidad de piston 2D a conjunto de particulas.
//------------------------------------------------------------------------------
template<byte periactive> __global__ void KerMovePiston2d(unsigned n,unsigned idini
  ,double dp,double posymin,double poszmin,unsigned poszcount,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(p<n){
    const unsigned id=p+idini;
    int pid=ridpmv[id];
    if(pid>=0){
      const double2 rpxy=posxy[pid];
      const double rpz=posz[pid];
      const unsigned cy=unsigned((rpxy.y-posymin)/dp);
      const unsigned cz=unsigned((rpz-poszmin)/dp);
      const double rmovx=(cz<poszcount? movx[cy*poszcount+cz]: 0);
      const float rvelx=float(cz<poszcount? velx[cy*poszcount+cz]: 0);
      //-Actualiza posicion.
      KerUpdatePos<periactive>(rpxy,rpz,rmovx,0,0,false,pid,posxy,posz,dcell,code);
      //-Actualiza velocidad.
      velrhop[pid].x=rvelx;
    }
  }
}

//==============================================================================
/// Applies movement and velocity of piston 2D to a group of particles.
/// Aplica movimiento y velocidad de piston 2D a conjunto de particulas.
//==============================================================================
void MovePiston2d(bool periactive,unsigned np,unsigned idini
  ,double dp,double posymin,double poszmin,unsigned poszcount,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerMovePiston2d<true>  <<<sgrid,SPHBSIZE>>> (np,idini,dp,posymin,poszmin,poszcount,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMovePiston2d<false> <<<sgrid,SPHBSIZE>>> (np,idini,dp,posymin,poszmin,poszcount,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
  }
}


//##############################################################################
//# Kernels for Floating bodies.
//##############################################################################
//==============================================================================
/// Computes distance between floating and centre particles according to periodic conditions.
/// Calcula distancia entre pariculas floating y centro segun condiciones periodicas.
//==============================================================================
template<bool periactive> __device__ void KerFtPeriodicDist(double px,double py,double pz,double cenx,double ceny,double cenz,float radius,float &dx,float &dy,float &dz){
  if(periactive){
    double ddx=px-cenx;
    double ddy=py-ceny;
    double ddz=pz-cenz;
    const unsigned peri=CTE.periactive;
    if(PERI_AxisX(peri) && fabs(ddx)>radius){
      if(ddx>0){ ddx+=CTE.xperincx; ddy+=CTE.xperincy; ddz+=CTE.xperincz; }
      else{      ddx-=CTE.xperincx; ddy-=CTE.xperincy; ddz-=CTE.xperincz; }
    }
    if(PERI_AxisY(peri) && fabs(ddy)>radius){
      if(ddy>0){ ddx+=CTE.yperincx; ddy+=CTE.yperincy; ddz+=CTE.yperincz; }
      else{      ddx-=CTE.yperincx; ddy-=CTE.yperincy; ddz-=CTE.yperincz; }
    }
    if(PERI_AxisZ(peri) && fabs(ddz)>radius){
      if(ddz>0){ ddx+=CTE.zperincx; ddy+=CTE.zperincy; ddz+=CTE.zperincz; }
      else{      ddx-=CTE.zperincx; ddy-=CTE.zperincy; ddz-=CTE.zperincz; }
    }
    dx=float(ddx);
    dy=float(ddy);
    dz=float(ddz);
  }
  else{
    dx=float(px-cenx);
    dy=float(py-ceny);
    dz=float(pz-cenz);
  }
}

//------------------------------------------------------------------------------
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtCalcForcesSum( //ftodatp={pini,np,radius,massp}
  const float4 *ftodatp,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  extern __shared__ float rfacex[];
  float *rfacey=rfacex+blockDim.x;
  float *rfacez=rfacey+blockDim.x;
  float *rfomegaacex=rfacez+blockDim.x;
  float *rfomegaacey=rfomegaacex+blockDim.x;
  float *rfomegaacez=rfomegaacey+blockDim.x;

  const unsigned tid=threadIdx.x;  //-Thread number.
  const unsigned cf=blockIdx.x;    //-Floating number.
  
  //-Loads floating data.
  const float4 rfdata=ftodatp[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  const float fmassp=rfdata.w;
  const double3 rcenter=ftocenter[cf];

  //-Initialises shared memory to zero.
  const unsigned ntid=(fnp<blockDim.x? fnp: blockDim.x); //-Number of used threads. | Numero de threads utilizados.
  if(tid<ntid){
    rfacex[tid]=rfacey[tid]=rfacez[tid]=0;
    rfomegaacex[tid]=rfomegaacey[tid]=rfomegaacez[tid]=0;
  }

  //-Computes data in shared memory. | Calcula datos en memoria shared.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned p=cfor*blockDim.x+tid;
    if(p<fnp){
      const unsigned rp=ftridp[p+fpini];
      if(rp!=UINT_MAX){
        float3 force=ace[rp];
        force.x*=fmassp; force.y*=fmassp; force.z*=fmassp;
        rfacex[tid]+=force.x; rfacey[tid]+=force.y; rfacez[tid]+=force.z;
        //-Computes distance from the centre. | Calcula distancia al centro.
        double2 rposxy=posxy[rp];
        float dx,dy,dz;
        KerFtPeriodicDist<periactive>(rposxy.x,rposxy.y,posz[rp],rcenter.x,rcenter.y,rcenter.z,fradius,dx,dy,dz);
        //-Computes omegaace.
        rfomegaacex[tid]+=(force.z*dy - force.y*dz);
        rfomegaacey[tid]+=(force.x*dz - force.z*dx);
        rfomegaacez[tid]+=(force.y*dx - force.x*dy);
      }
    }
  }

  //-Reduces data in shared memory and stores results.
  //-Reduce datos de memoria shared y guarda resultados.
  __syncthreads();
  if(!tid){
    float3 face=make_float3(0,0,0);
    float3 fomegaace=make_float3(0,0,0);
    for(unsigned c=0;c<ntid;c++){
      face.x+=rfacex[c];  face.y+=rfacey[c];  face.z+=rfacez[c];
      fomegaace.x+=rfomegaacex[c]; fomegaace.y+=rfomegaacey[c]; fomegaace.z+=rfomegaacez[c];
    }
    //-Stores results in ftoforcessum[].
    unsigned cf2=cf*2;
    float3 aux=ftoforcessum[cf2];
    face.x+=aux.x; face.y+=aux.y; face.z+=aux.z;
    ftoforcessum[cf2]=face;
    cf2++;
    aux=ftoforcessum[cf2];
    fomegaace.x+=aux.x; fomegaace.y+=aux.y; fomegaace.z+=aux.z;
    ftoforcessum[cf2]=fomegaace;
  }
}

//==============================================================================
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//==============================================================================
void FtCalcForcesSum(bool periactive,unsigned ftcount
  ,const float4 *ftodatp,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  if(ftcount){
    const unsigned bsize=256;
    const unsigned smem=sizeof(float)*(3+3)*bsize;
    dim3 sgrid=GetSimpleGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtCalcForcesSum<true>  <<<sgrid,bsize,smem>>> (ftodatp,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
    else          KerFtCalcForcesSum<false> <<<sgrid,bsize,smem>>> (ftodatp,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
  }
}

//------------------------------------------------------------------------------
/// Carga valores de matriz 3x3 en bloques de 4, 4 y 1.
/// Loads values of matrix 3x3 in blocks of 4, 4 y 1.
//------------------------------------------------------------------------------
__device__ void KerLoadMatrix3f(unsigned c,const float4 *data8,const float *data1,tmatrix3f &v)
{
  float4 v4=data8[c*2];
  v.a11=v4.x; v.a12=v4.y; v.a13=v4.z; v.a21=v4.w;
  v4=data8[c*2+1];
  v.a22=v4.x; v.a23=v4.y; v.a31=v4.z; v.a32=v4.w;
  v.a33=data1[c];
}

//------------------------------------------------------------------------------
/// Computes final acceleration from particles and from external forces to ftoforces[].
/// Calcula aceleracion final a parti de particulas y de fuerzas externas en ftoforces[].
//------------------------------------------------------------------------------
__global__ void KerFtCalcForces(unsigned ftcount,float3 gravity
  ,const float *ftomass,const float3 *ftoangles
  ,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,float3 *ftoforces) //fdata={pini,np,radius,mass}
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Number of floating.
  if(cf<ftcount){
    //-Loads floating data.
    const float fmass=ftomass[cf];
    const float3 fang=ftoangles[cf];
    tmatrix3f inert;
    KerLoadMatrix3f(cf,ftoinertiaini8,ftoinertiaini1,inert);

    //-Compute a cumulative rotation matrix.
    const tmatrix3f frot=cumath::RotMatrix3x3(fang);
    //-Compute the inertia tensor by rotating the initial tensor to the curent orientation I=(R*I_0)*R^T.
    inert=cumath::MulMatrix3x3(cumath::MulMatrix3x3(frot,inert),cumath::TrasMatrix3x3(frot));
    //-Calculates the inverse of the inertia matrix to compute the I^-1 * L= W
    const tmatrix3f invinert=cumath::InverseMatrix3x3(inert);

    //-Loads traslational and rotational velocities.
    const unsigned cf2=cf*2;
    float3 face=ftoforces[cf2];
    float3 fomegaace=ftoforces[cf2+1];

    //-Calculate omega starting from fomegaace & invinert. | Calcula omega a partir de fomegaace y invinert.
    {
      float3 omegaace;
      omegaace.x=(fomegaace.x*invinert.a11+fomegaace.y*invinert.a12+fomegaace.z*invinert.a13);
      omegaace.y=(fomegaace.x*invinert.a21+fomegaace.y*invinert.a22+fomegaace.z*invinert.a23);
      omegaace.z=(fomegaace.x*invinert.a31+fomegaace.y*invinert.a32+fomegaace.z*invinert.a33);
      fomegaace=omegaace;
    }
    //-Add gravity force and divide by mass. | Suma fuerza de gravedad y divide por la masa.
    face.x=(face.x + fmass*gravity.x) / fmass;
    face.y=(face.y + fmass*gravity.y) / fmass;
    face.z=(face.z + fmass*gravity.z) / fmass;
    //-Stores final results.
    ftoforces[cf2]  =face; //-Saves acceleration (forces/fmass);
    ftoforces[cf2+1]=fomegaace;
  }
}

//==============================================================================
/// Computes final acceleration from particles and from external forces to ftoforces[].
/// Calcula aceleracion final a parti de particulas y de fuerzas externas en ftoforces[].
//==============================================================================
void FtCalcForces(unsigned ftcount,tfloat3 gravity
  ,const float *ftomass,const float3 *ftoangles
  ,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,float3 *ftoforces)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtCalcForces <<<sgrid,SPHBSIZE>>> (ftcount,Float3(gravity),ftomass
      ,ftoangles,ftoinertiaini8,ftoinertiaini1,ftoforces);
  }
}


//------------------------------------------------------------------------------
/// Calculate data to update floatings.
/// Calcula datos para actualizar floatings.
//------------------------------------------------------------------------------
__global__ void KerFtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftovelace,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
  if(cf<ftcount){
    //-Compute fomega.
    float3 fomega=ftovelace[ftcount+cf];
    {
      const float3 omegaace=ftoforces[cf*2+1];
      fomega.x=float(dt*omegaace.x+fomega.x);
      fomega.y=float(dt*omegaace.y+fomega.y);
      fomega.z=float(dt*omegaace.z+fomega.z);
    }
    float3 fvel=ftovelace[cf];
    //-Zero components for 2-D simulation. | Anula componentes para 2D.
    float3 face=ftoforces[cf*2];
    if(simulate2d){ face.y=0; fomega.x=0; fomega.z=0; fvel.y=0; }
    //-Compute fcenter.
    double3 fcenter=ftocenter[cf];
    fcenter.x+=dt*fvel.x;
    fcenter.y+=dt*fvel.y;
    fcenter.z+=dt*fvel.z;
    //-Compute fvel.
    fvel.x=float(dt*face.x+fvel.x);
    fvel.y=float(dt*face.y+fvel.y);
    fvel.z=float(dt*face.z+fvel.z);
    //-Store data to update floating. | Guarda datos para actualizar floatings.
    ftoforcesres[cf*2]=fomega;
    ftoforcesres[cf*2+1]=fvel;
    ftocenterres[cf]=fcenter;
  }
}

//==============================================================================
/// Computes forces on floatings.
/// Calcula fuerzas sobre floatings.
//==============================================================================
void FtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftovelace,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtCalcForcesRes <<<sgrid,SPHBSIZE>>> (ftcount,simulate2d,dt,ftovelace,ftocenter,ftoforces,ftoforcesres,ftocenterres);
  }
}


//------------------------------------------------------------------------------
/// Applies motion constraints.
/// Aplica restricciones de movimiento.
//------------------------------------------------------------------------------
__global__ void KerFtApplyConstraints(unsigned ftcount,const byte *ftoconstraints
  ,float3 *ftoforces,float3 *ftoforcesres)
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
  if(cf<ftcount){
    //-Applies motion constraints.
    const byte constr=ftoconstraints[cf];
    if(constr!=0){
      const unsigned cf2=cf*2;
      const unsigned cf21=cf2+1;
      float3 face=ftoforces[cf2];
      float3 fomegaace=ftoforces[cf21];
      float3 fomega=ftoforcesres[cf2];
      float3 fvel=ftoforcesres[cf21];
      //-Updates values.
      face.x=(constr&FTCON_MoveX? 0: face.x);
      face.y=(constr&FTCON_MoveY? 0: face.y);
      face.z=(constr&FTCON_MoveZ? 0: face.z);
      fomegaace.x=(constr&FTCON_RotateX? 0: fomegaace.x);
      fomegaace.y=(constr&FTCON_RotateY? 0: fomegaace.y);
      fomegaace.z=(constr&FTCON_RotateZ? 0: fomegaace.z);
      fvel.x=(constr&FTCON_MoveX? 0: fvel.x);
      fvel.y=(constr&FTCON_MoveY? 0: fvel.y);
      fvel.z=(constr&FTCON_MoveZ? 0: fvel.z);
      fomega.x=(constr&FTCON_RotateX? 0: fomega.x);
      fomega.y=(constr&FTCON_RotateY? 0: fomega.y);
      fomega.z=(constr&FTCON_RotateZ? 0: fomega.z);
      //-Stores updated values.
      ftoforces[cf2]=face;
      ftoforces[cf21]=fomegaace;
      ftoforcesres[cf2]=fomega;
      ftoforcesres[cf21]=fvel;
    }
  }
}

//==============================================================================
/// Applies motion constraints.
/// Aplica restricciones de movimiento.
//==============================================================================
void FtApplyConstraints(unsigned ftcount,const byte *ftoconstraints
  ,float3 *ftoforces,float3 *ftoforcesres)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtApplyConstraints <<<sgrid,SPHBSIZE>>> (ftcount,ftoconstraints,ftoforces,ftoforcesres);
  }
}


//------------------------------------------------------------------------------
/// Updates information and particles of floating bodies.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtUpdate(bool predictor,double dt //ftodata={pini,np,radius,massp}
  ,unsigned nft,const float4 *ftodatp,const float3 *ftoforcesres
  ,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovelace
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  const unsigned tid=threadIdx.x;  //-Thread number.
  const unsigned cf=blockIdx.x;    //-Floating number.
  //-Obtains floating data.
  const float3 fomega=ftoforcesres[cf*2];
  const float3 fvel=ftoforcesres[cf*2+1];
  const double3 fcenter=ftocenterres[cf];
  float4 rfdata=ftodatp[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  //-Updates floating particles.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned fp=cfor*blockDim.x+tid;
    if(fp<fnp){
      const unsigned p=ftridp[fp+fpini];
      if(p!=UINT_MAX){
        double2 rposxy=posxy[p];
        double rposz=posz[p];
        float4 rvel=velrhop[p];
        //-Computes and stores position displacement.
        const double dx=dt*double(rvel.x);
        const double dy=dt*double(rvel.y);
        const double dz=dt*double(rvel.z);
        KerUpdatePos<periactive>(rposxy,rposz,dx,dy,dz,false,p,posxy,posz,dcell,code);
        //-Computes and stores new velocity.
        float disx,disy,disz;
        KerFtPeriodicDist<periactive>(rposxy.x+dx,rposxy.y+dy,rposz+dz,fcenter.x,fcenter.y,fcenter.z,fradius,disx,disy,disz);
        rvel.x=fvel.x+(fomega.y*disz-fomega.z*disy);
        rvel.y=fvel.y+(fomega.z*disx-fomega.x*disz);
        rvel.z=fvel.z+(fomega.x*disy-fomega.y*disx);
        velrhop[p]=rvel;
      }
    }
  }

  //-Stores floating data.
  __syncthreads();
  if(!tid && !predictor){
    ftocenter[cf]=(periactive? KerUpdatePeriodicPos(fcenter): fcenter);
    float3 rangles=ftoangles[cf];
    rangles.x=float(double(rangles.x)+double(fomega.x)*dt);
    rangles.y=float(double(rangles.y)+double(fomega.y)*dt);
    rangles.z=float(double(rangles.z)+double(fomega.z)*dt);
    ftoangles[cf]=rangles;
    //-Linear velocity and acceleration.
    float3 v=ftovelace[cf];
    v.x=(fvel.x-v.x)/float(dt);
    v.y=(fvel.y-v.y)/float(dt);
    v.z=(fvel.z-v.z)/float(dt);
    ftovelace[cf]=fvel;
    ftovelace[nft+nft+cf]=v;
    //-Angular velocity and acceleration.
    v=ftovelace[nft+cf];
    v.x=(fomega.x-v.x)/float(dt);
    v.y=(fomega.y-v.y)/float(dt);
    v.z=(fomega.z-v.z)/float(dt);
    ftovelace[nft+cf]=fomega;
    ftovelace[nft*3+cf]=v;
  }
}

//==============================================================================
/// Updates information and particles of floating bodies.
//==============================================================================
void FtUpdate(bool periactive,bool predictor,unsigned ftcount,double dt
  ,const float4 *ftodatp,const float3 *ftoforcesres,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovelace
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(ftcount){
    const unsigned bsize=128; 
    dim3 sgrid=GetSimpleGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtUpdate<true>  <<<sgrid,bsize>>> (predictor,dt,ftcount,ftodatp,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovelace,posxy,posz,dcell,velrhop,code);
    else          KerFtUpdate<false> <<<sgrid,bsize>>> (predictor,dt,ftcount,ftodatp,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovelace,posxy,posz,dcell,velrhop,code);
  }
}


//<vs_ftmottionsv_ini>
//------------------------------------------------------------------------------
/// Get reference position of floating bodies.
//------------------------------------------------------------------------------
__global__ void KerFtGetPosRef(unsigned np,const unsigned *idpref
  ,const unsigned *ftridp,const double2 *posxy,const double *posz,double *posref)
{
  unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(cp<np){
    bool ok=false;
    const unsigned cid=idpref[cp];
    if(cid!=UINT_MAX){
      const unsigned p=ftridp[cid];
      if(p!=UINT_MAX){
        const double2 rxy=posxy[p];
        const unsigned c=cp*3;
        posref[c  ]=rxy.x;
        posref[c+1]=rxy.y;
        posref[c+2]=posz[p];
        ok=true;
      }
    }
    if(!ok)posref[cp*3]=DBL_MAX;
  }
}
//==============================================================================
/// Get reference position of floating bodies.
//==============================================================================
void FtGetPosRef(unsigned np,const unsigned *idpref,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,double *posref)
{
  if(np){
    const unsigned bsize=128; 
    dim3 sgrid=GetSimpleGridSize(np,bsize);
    KerFtGetPosRef <<<sgrid,bsize>>> (np,idpref,ftridp,posxy,posz,posref);
  }
}
//<vs_ftmottionsv_end>



//##############################################################################
//# Kernels for Periodic conditions
//# Kernels para Periodic conditions
//##############################################################################
//------------------------------------------------------------------------------
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicIgnore(unsigned n,typecode *code)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    //-Checks code of particles.
    //-Comprueba codigo de particula.
    const typecode rcode=code[p];
    if(CODE_IsPeriodic(rcode))code[p]=CODE_SetOutIgnore(rcode);
  }
}

//==============================================================================
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//==============================================================================
void PeriodicIgnore(unsigned n,typecode *code){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicIgnore <<<sgrid,SPHBSIZE>>> (n,code);
  }
}

//------------------------------------------------------------------------------
/// Create list of new periodic particles to be duplicated and 
/// marks old periodics to be ignored.
///
/// Crea lista de nuevas particulas periodicas a duplicar y con delper activado
/// marca las periodicas viejas para ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicMakeList(unsigned n,unsigned pini,unsigned nmax
  ,double3 mapposmin,double3 mapposmax,double3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p2=p+pini;
    //-Inteacts with normal or periodic particles.
    //-Se queda con particulas normales o periodicas.
    if(CODE_GetSpecialValue(code[p2])<=CODE_PERIODIC){
      //-Obtains particle position.
      const double2 rxy=posxy[p2];
      const double rx=rxy.x,ry=rxy.y;
      const double rz=posz[p2];
      double rx2=rx+perinc.x,ry2=ry+perinc.y,rz2=rz+perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=p2;
      }
      rx2=rx-perinc.x; ry2=ry-perinc.y; rz2=rz-perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=(p2|0x80000000);
      }
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    unsigned cp=slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[threadIdx.x+1];
  }
  if(blockDim.x+threadIdx.x<ns){ //-There may be twice as many periodics per thread. | Puede haber el doble de periodicas que threads.
    unsigned cp=blockDim.x+slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[blockDim.x+threadIdx.x+1];
  }
}

//==============================================================================
/// Create list of new periodic particles to be duplicated.
/// With stable activated reorders perioc list.
///
/// Crea lista de nuevas particulas periodicas a duplicar.
/// Con stable activado reordena lista de periodicas.
//==============================================================================
unsigned PeriodicMakeList(unsigned n,unsigned pini,bool stable,unsigned nmax
  ,tdouble3 mapposmin,tdouble3 mapposmax,tdouble3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-lspg size list initialized to zero.
    //-Inicializa tamanho de lista lspg a cero.
    hipMemset(listp+nmax,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE*2+1)*sizeof(unsigned); //-Each particle can leave two new periodic over the counter position. | De cada particula pueden salir 2 nuevas periodicas mas la posicion del contador.
    KerPeriodicMakeList <<<sgrid,SPHBSIZE,smem>>> (n,pini,nmax,Double3(mapposmin),Double3(mapposmax),Double3(perinc),posxy,posz,code,listp);
    hipMemcpy(&count,listp+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list(listp);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}

//------------------------------------------------------------------------------
/// Doubles the position of the indicated particle using a displacement.
/// Duplicate particles are considered valid and are always within
/// the domain.
/// This kernel applies to single-GPU and multi-GPU because the calculations are made
/// from domposmin.
/// It controls the cell coordinates not exceed the maximum.
///
/// Duplica la posicion de la particula indicada aplicandole un desplazamiento.
/// Las particulas duplicadas se considera que siempre son validas y estan dentro
/// del dominio.
/// Este kernel vale para single-gpu y multi-gpu porque los calculos se hacen 
/// a partir de domposmin.
/// Se controla que las coordendas de celda no sobrepasen el maximo.
//------------------------------------------------------------------------------
__device__ void KerPeriodicDuplicatePos(unsigned pnew,unsigned pcopy
  ,bool inverse,double dx,double dy,double dz,uint3 cellmax
  ,double2 *posxy,double *posz,unsigned *dcell)
{
  //-Obtains position of the particle to be duplicated.
  //-Obtiene pos de particula a duplicar.
  double2 rxy=posxy[pcopy];
  double rz=posz[pcopy];
  //-Applies displacement.
  rxy.x+=(inverse? -dx: dx);
  rxy.y+=(inverse? -dy: dy);
  rz+=(inverse? -dz: dz);
  //-Computes cell coordinates within the domain.
  //-Calcula coordendas de celda dentro de dominio.
  unsigned cx=unsigned((rxy.x-CTE.domposminx)/CTE.scell);
  unsigned cy=unsigned((rxy.y-CTE.domposminy)/CTE.scell);
  unsigned cz=unsigned((rz-CTE.domposminz)/CTE.scell);
  //-Adjust cell coordinates if they exceed the maximum.
  //-Ajusta las coordendas de celda si sobrepasan el maximo.
  cx=(cx<=cellmax.x? cx: cellmax.x);
  cy=(cy<=cellmax.y? cy: cellmax.y);
  cz=(cz<=cellmax.z? cz: cellmax.z);
  //-Stores position and cell of the new particles.
  //-Graba posicion y celda de nuevas particulas.
  posxy[pnew]=rxy;
  posz[pnew]=rz;
  dcell[pnew]=DCEL_Cell(CTE.cellcode,cx,cy,cz);
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
__global__ void KerPeriodicDuplicateVerlet(unsigned n,unsigned pini,uint3 cellmax,double3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    velrhopm1[pnew]=velrhopm1[pcopy];
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateVerlet(unsigned n,unsigned pini,tuint3 domcells,tdouble3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateVerlet <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,velrhopm1);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
template<bool varspre> __global__ void KerPeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,uint3 cellmax,double3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    if(varspre){
      posxypre[pnew]=posxypre[pcopy];
      poszpre[pnew]=poszpre[pcopy];
      velrhoppre[pnew]=velrhoppre[pcopy];
    }
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,tuint3 domcells,tdouble3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    if(posxypre!=NULL)KerPeriodicDuplicateSymplectic<true>  <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
    else              KerPeriodicDuplicateSymplectic<false> <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
__global__ void KerPeriodicDuplicateNormals(unsigned n,unsigned pini,const unsigned *listp,float3 *normals,float3 *motionvel)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    normals[pnew]=normals[pcopy];
    if(motionvel)motionvel[pnew]=motionvel[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateNormals(unsigned n,unsigned pini,const unsigned *listp,float3 *normals,float3 *motionvel)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateNormals <<<sgrid,SPHBSIZE>>> (n,pini,listp,normals,motionvel);
  }
}

//##############################################################################
//# Kernels for Damping.
//##############################################################################
//------------------------------------------------------------------------------
/// Returns TRUE when code==NULL or particle is normal and fluid.
//------------------------------------------------------------------------------
__device__ bool KerIsNormalFluid(const typecode *code,unsigned p){
  if(code){//-Descarta particulas floating o periodicas.
    const typecode cod=code[p];
    return(CODE_IsNormal(cod) && CODE_IsFluid(cod));
  }
  return(true);
}
//------------------------------------------------------------------------------
/// Checks position is inside box limits.
/// Comprueba si la posicion esta dentro de los limites.
//------------------------------------------------------------------------------
__device__ bool KerPointInBox(double px,double py,double pz,const double3 &p1,const double3 &p2)
{
  return(p1.x<=px && p1.y<=py && p1.z<=pz && px<=p2.x && py<=p2.y && pz<=p2.z);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,double px,double py,double pz)
{
  return(pla.x*px+pla.y*py+pla.z*pz+pla.w);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,const double3 &pt)
{
  return(pla.x*pt.x+pla.y*pt.y+pla.z*pt.z+pla.w);
}

//------------------------------------------------------------------------------
/// Applies Damping.
/// Aplica Damping.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingPlane(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      double vdis=KerPointPlane(plane,rposxy.x,rposxy.y,rposz);  //fgeo::PlanePoint(plane,ps);
      if(0<vdis && vdis<=dist+over){
        const double fdis=(vdis>=dist? 1.: vdis/dist);
        const double redudt=dt*(fdis*fdis)*redumax;
        double redudtx=(1.-redudt*factorxyz.x);
        double redudty=(1.-redudt*factorxyz.y);
        double redudtz=(1.-redudt*factorxyz.z);
        redudtx=(redudtx<0? 0.: redudtx);
        redudty=(redudty<0? 0.: redudty);
        redudtz=(redudtz<0? 0.: redudtz);
        float4 rvel=velrhop[p1];
        rvel.x=float(redudtx*rvel.x); 
        rvel.y=float(redudty*rvel.y); 
        rvel.z=float(redudtz*rvel.z);
        velrhop[p1]=rvel;
      }
    }
  }
}
//==============================================================================
/// Applies Damping.
/// Aplica Damping.
//==============================================================================
void ComputeDampingPlane(double dt,double4 plane,float dist,float over
  ,float3 factorxyz,float redumax,unsigned n,unsigned pini
  ,const double2 *posxy,const double *posz,const typecode *code,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDampingPlane <<<sgridf,SPHBSIZE>>> (n,pini,dt,plane,dist,over
      ,factorxyz,redumax,posxy,posz,code,velrhop);
  }
}

//------------------------------------------------------------------------------
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingPlaneDom(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,double zmin,double zmax,double4 pla0,double4 pla1,double4 pla2,double4 pla3
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      const double3 ps=make_double3(rposxy.x,rposxy.y,rposz);
      double vdis=KerPointPlane(plane,ps);  //fgeo::PlanePoint(plane,ps);
      if(0<vdis && vdis<=dist+over){
        if(ps.z>=zmin && ps.z<=zmax && KerPointPlane(pla0,ps)<=0 && KerPointPlane(pla1,ps)<=0 && KerPointPlane(pla2,ps)<=0 && KerPointPlane(pla3,ps)<=0){
          const double fdis=(vdis>=dist? 1.: vdis/dist);
          const double redudt=dt*(fdis*fdis)*redumax;
          double redudtx=(1.-redudt*factorxyz.x);
          double redudty=(1.-redudt*factorxyz.y);
          double redudtz=(1.-redudt*factorxyz.z);
          redudtx=(redudtx<0? 0.: redudtx);
          redudty=(redudty<0? 0.: redudty);
          redudtz=(redudtz<0? 0.: redudtz);
          float4 rvel=velrhop[p1];
          rvel.x=float(redudtx*rvel.x); 
          rvel.y=float(redudty*rvel.y); 
          rvel.z=float(redudtz*rvel.z); 
          velrhop[p1]=rvel;
        }
      }
    }
  }
}
//==============================================================================
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//==============================================================================
void ComputeDampingPlaneDom(double dt,double4 plane,float dist,float over
  ,float3 factorxyz,float redumax
  ,double zmin,double zmax,double4 pla0,double4 pla1,double4 pla2,double4 pla3
  ,unsigned n,unsigned pini,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDampingPlaneDom <<<sgridf,SPHBSIZE>>> (n,pini,dt,plane,dist,over,factorxyz
      ,redumax,zmin,zmax,pla0,pla1,pla2,pla3,posxy,posz,code,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Applies Damping according box configuration.
/// Aplica Damping segun cofiguracion de caja.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingBox(unsigned n,unsigned pini
  ,double dt,float3 factorxyz,float redumax
  ,double3 limitmin1,double3 limitmin2,double3 limitmax1,double3 limitmax2
  ,double3 limitover1,double3 limitover2,double3 boxsize1,double3 boxsize2
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      //-Check if it is within the domain. | Comprueba si esta dentro del dominio.
      if(KerPointInBox(rposxy.x,rposxy.y,rposz,limitover1,limitover2)){//-Inside overlimit domain.
        if(!KerPointInBox(rposxy.x,rposxy.y,rposz,limitmin1,limitmin2)){//-Outside free domain.
          double fdis=1.;
          if(KerPointInBox(rposxy.x,rposxy.y,rposz,limitmax1,limitmax2)){//-Compute damping coefficient.
            fdis=0;
            if(boxsize2.z){ const double fdiss=(rposz   -limitmin2.z)/boxsize2.z; fdis=(fdis>=fdiss? fdis: fdiss); }
            if(boxsize2.y){ const double fdiss=(rposxy.y-limitmin2.y)/boxsize2.y; fdis=(fdis>=fdiss? fdis: fdiss); }
            if(boxsize2.x){ const double fdiss=(rposxy.x-limitmin2.x)/boxsize2.x; fdis=(fdis>=fdiss? fdis: fdiss); }
            if(boxsize1.z){ const double fdiss=(limitmin1.z-rposz   )/boxsize1.z; fdis=(fdis>=fdiss? fdis: fdiss); }
            if(boxsize1.y){ const double fdiss=(limitmin1.y-rposxy.y)/boxsize1.y; fdis=(fdis>=fdiss? fdis: fdiss); }
            if(boxsize1.x){ const double fdiss=(limitmin1.x-rposxy.x)/boxsize1.x; fdis=(fdis>=fdiss? fdis: fdiss); }
          }
          const double redudt=dt*(fdis*fdis)*redumax;
          double redudtx=(1.-redudt*factorxyz.x);
          double redudty=(1.-redudt*factorxyz.y);
          double redudtz=(1.-redudt*factorxyz.z);
          redudtx=(redudtx<0? 0.: redudtx);
          redudty=(redudty<0? 0.: redudty);
          redudtz=(redudtz<0? 0.: redudtz);
          float4 rvel=velrhop[p1];
          rvel.x=float(redudtx*rvel.x); 
          rvel.y=float(redudty*rvel.y); 
          rvel.z=float(redudtz*rvel.z);
          //rvel.x=rvel.y=rvel.z=0;
          velrhop[p1]=rvel;
        }
      }
    }
  }
}
//==============================================================================
/// Applies Damping according box configuration.
/// Aplica Damping segun cofiguracion de caja.
//==============================================================================
void ComputeDampingBox(unsigned n,unsigned pini,double dt,float3 factorxyz,float redumax
  ,double3 limitmin1,double3 limitmin2,double3 limitmax1,double3 limitmax2
  ,double3 limitover1,double3 limitover2,double3 boxsize1,double3 boxsize2
  ,const double2 *posxy,const double *posz,const typecode *code,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDampingBox <<<sgridf,SPHBSIZE>>> (n,pini,dt,factorxyz,redumax
      ,limitmin1,limitmin2,limitmax1,limitmax2,limitover1,limitover2,boxsize1,boxsize2
      ,posxy,posz,code,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Applies Damping to limited cylinder domain.
/// Aplica Damping limitado a un dominio de cilindro.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingCylinder(unsigned n,unsigned pini
  ,double dt,bool isvertical,double3 point1,double3 point2,double limitmin
  ,float dist,float over,float3 factorxyz,float redumax
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      //-Check if it is within the domain. | Comprueba si esta dentro del dominio.
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      const double3 ps=make_double3(rposxy.x,rposxy.y,rposz);
      const double vdis=(isvertical? 
        sqrt((ps.x-point1.x)*(ps.x-point1.x)+(ps.y-point1.y)*(ps.y-point1.y)): 
        cugeo::LinePointDist(ps,point1,point2)
        ) - limitmin;
      if(0<vdis && vdis<=dist+over){
        const double fdis=(vdis>=dist? 1.: vdis/dist);
        const double redudt=dt*(fdis*fdis)*redumax;
        double redudtx=(1.-redudt*factorxyz.x);
        double redudty=(1.-redudt*factorxyz.y);
        double redudtz=(1.-redudt*factorxyz.z);
        redudtx=(redudtx<0? 0.: redudtx);
        redudty=(redudty<0? 0.: redudty);
        redudtz=(redudtz<0? 0.: redudtz);
        float4 rvel=velrhop[p1];
        rvel.x=float(redudtx*rvel.x); 
        rvel.y=float(redudty*rvel.y); 
        rvel.z=float(redudtz*rvel.z); 
        velrhop[p1]=rvel;
      }
    }
  }
}
//==============================================================================
/// Applies Damping to limited cylinder domain.
/// Aplica Damping limitado a un dominio de cilindro.
//==============================================================================
void ComputeDampingCylinder(unsigned n,unsigned pini
  ,double dt,double3 point1,double3 point2,double limitmin
  ,float dist,float over,float3 factorxyz,float redumax
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    const bool isvertical=(point1.x==point2.x && point1.y==point2.y);
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDampingCylinder <<<sgridf,SPHBSIZE>>> (n,pini,dt
      ,isvertical,point1,point2,limitmin,dist,over,factorxyz,redumax
      ,posxy,posz,code,velrhop);
  }
}


}


//##############################################################################
//# Kernels for InOut (JSphInOut).
//# Kernels para InOut (JSphInOut).
//##############################################################################
#include "JSphGpu_InOut_iker.cu"


